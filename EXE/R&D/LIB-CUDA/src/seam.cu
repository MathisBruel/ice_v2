#include "hip/hip_runtime.h"
#include <seam.cuh>

__global__ void cutFrame(int width, int height, int sizeBuffer, unsigned char* dataRGB, unsigned char* left, unsigned char* right)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int idx = index; idx < width*height*3*sizeBuffer; idx+=stride) {

        int comp = idx%3;
        int idxTemp = (idx-comp)/3;
        int i = idxTemp%width;
        int tmp = (idxTemp-i)/width;
        int j = tmp%height;
        int frame = (tmp - j)/height;

        int tempWidth = width/2;

        if (i < tempWidth) {
            left[((frame*height+j)*tempWidth+i)*3+comp] = dataRGB[idx];
        }
        else {
            right[(((frame*height+j)*tempWidth)+i-tempWidth)*3+comp] = dataRGB[idx];
        }
    }
}
__global__ void copyImage(int width, int height, int sizeBuffer, unsigned char* dataInLeft, unsigned char* dataInRight, unsigned char* dataOut) 
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int idx = index; idx < width*height*3*sizeBuffer; idx+=stride) {
        
        unsigned char comp = idx%3;
        int idxTemp = (idx-comp)/3;
        short i = idxTemp%width;
        int tmp = (idxTemp-i)/width;
        int j = tmp%height;
        int frame = (tmp - j)/height;
        int tempWidth = width/2;

        if (i < tempWidth) {
            dataOut[idx] = dataInLeft[(((frame*height+j)*tempWidth)+i)*3+comp];
        }
        else {
            dataOut[idx] = dataInRight[(((frame*height+j)*tempWidth)+i-tempWidth)*3+comp];
        }
    }
}

__global__ void grayscale(int width, int height, int sizeBuffer, unsigned char* dataRGB, float* dataOutGray)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < width*height*sizeBuffer; i+=stride) {
        dataOutGray[i] = (float)(dataRGB[i*3])*0.2989f + (float)(dataRGB[i*3+1])*0.587f + (float)(dataRGB[i*3+2])*0.114f;
        dataOutGray[i] /= 255;
    }
}
__global__ void sobel(int width, int height, int sizeBuffer, float* dataInGray, float* dataOutSobel)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int idx = index; idx < height*width*sizeBuffer; idx+=stride) {

        int i = idx%width;
        int j = (idx - i)/width;
        j %= height;

        float color1 = (i == 0 || j == 0) ? 0 : dataInGray[idx - width - 1];
        float color2 = (j == 0) ? 0 : dataInGray[idx - width];
        float color3 = (i == width-1 || j == 0) ? 0 :  dataInGray[idx - width + 1];
        float color4 = (i == 0) ? 0 : dataInGray[idx - 1];
        float color5 = (i == width-1) ? 0 : dataInGray[idx + 1];
        float color6 = (i == 0 || j == height-1) ? 0 : dataInGray[idx + width - 1];
        float color7 = (j == height-1) ? 0 : dataInGray[idx + width];
        float color8 = (i == width-1 || j == height-1) ? 0 : dataInGray[idx + width + 1];

        float dx = color3 + 2*color5 + color8 - color1 - 2*color4 - color6;
        float dy = color6 + 2*color7 + color8 - color1 - 2*color2 - color3;
        dataOutSobel[idx] = sqrt(dx*dx + dy*dy);
    }
}
__global__ void mergeSobel(int width, int height, int frame, int size, float* dataInSobel, float* dataOutSobel)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int idx = index; idx < height*width; idx+=stride) {
        if (frame == 0) {
            dataOutSobel[idx] += dataInSobel[frame*width*height+idx]/size;
        }
        else {
            dataOutSobel[idx] += (fabs(dataInSobel[frame*width*height+idx] - dataInSobel[(frame-1)*width*height+idx])*0.7 + dataInSobel[frame*width*height+idx]*0.3)/size;
        }
    }
}

__global__ void removeSeamFromSobel(int width, int height, float *dataSobel, float *dataSobelNew, int* seam)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int idx = index; idx < width*height; idx+=stride) {

        int i = idx%width;
        int j = (idx-i)/width;
        int idx2 = j*(width+1)+i;

        if (i < seam[j]) {dataSobelNew[idx] = dataSobel[idx2];}
        else {dataSobelNew[idx] = dataSobel[idx2+1];}
    }
}
__global__ void addSeamToImage(int width, int height, int sizeBuffer, unsigned char* dataIn, unsigned char* dataOut, int* seam)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int idx = index; idx < width*height*3*sizeBuffer; idx+=stride) {

        int comp = idx%3;
        int idxTemp = (idx-comp)/3;
        int i = idxTemp%width;
        int tmp = (idxTemp-i)/width;
        int j = tmp%height;
        int frame = (tmp - j)/height;
        int idx2 = ((frame*height+j)*(width-1)+i)*3+comp;

        // -- before seam
        if (i <= seam[j]) {dataOut[idx] = dataIn[idx2];}
        // -- after seam
        else if (i > seam[j] + 1) {dataOut[idx] = dataIn[idx2-3];}
        // -- duplicate seam
        else {
            int value = dataIn[idx2-3];
            if (seam[j]+1 < width-1) {
                value += dataIn[idx2];
            }
            value /= 2;
            dataOut[idx] = (unsigned char)value;
        }
    }
}
__global__ void repercuteSeam(int nbSeamsImpact, int* seams, int height)
{
    for (int i = 0; i < nbSeamsImpact; i++) {
        for (int j = i+1; j < nbSeamsImpact; j++) {

            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int idx = index; idx < height; idx+=stride) {
        
                if (seams[j*height+idx] >= seams[i*height+idx]) {
                    seams[j*height+idx] += 2;
                }
            }
        }
    }
}

__global__ void computeIndexMap(int width, int height, float* sobelImage, int* indexes)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int idx = index; idx < width*(height-1); idx+=stride) {

        int i = idx%width;
        int tempIdx = idx+width;
        if (i == 0) {
            float middle = sobelImage[tempIdx];
            float right = sobelImage[tempIdx+1];

            if (middle <= right) {indexes[idx] = tempIdx;}
            else {indexes[idx] = tempIdx+1;}
        }
        else if (i == width-1) {
            float left = sobelImage[tempIdx-1];
            float middle = sobelImage[tempIdx];

            if (middle <= left) {indexes[idx] = tempIdx;}
            else {indexes[idx] = tempIdx-1;}
        }
        else {
            float left = sobelImage[tempIdx-1];
            float middle = sobelImage[tempIdx];
            float right = sobelImage[tempIdx+1];

            if (middle <= left && middle <= right) {indexes[idx] = tempIdx;}
            else if (left <= right) {indexes[idx] = tempIdx-1;}
            else {indexes[idx] = tempIdx+1;}
        }
    }
}
__global__ void computeIndexMap2(int width, int height, float* sobelImage, int* indexes)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int idx = index; idx < width*(height-1); idx+=stride) {

        int i = idx%width;
        int tempIdx = idx;
        if (i == 0) {
            float middle = sobelImage[tempIdx];
            float right = sobelImage[tempIdx+1];

            if (middle <= right) {indexes[idx] = tempIdx;}
            else {indexes[idx] = tempIdx+1;}
        }
        else if (i == width-1) {
            float left = sobelImage[tempIdx-1];
            float middle = sobelImage[tempIdx];

            if (middle <= left) {indexes[idx] = tempIdx;}
            else {indexes[idx] = tempIdx-1;}
        }
        else {
            float left = sobelImage[tempIdx-1];
            float middle = sobelImage[tempIdx];
            float right = sobelImage[tempIdx+1];

            if (middle <= left && middle <= right) {indexes[idx] = tempIdx;}
            else if (left <= right) {indexes[idx] = tempIdx-1;}
            else {indexes[idx] = tempIdx+1;}
        }
    }
}

__global__ void calculateAllSeams(int width, int height, int* indexes, float* sobel, int* seamsTemporaries, float* energy, int leftBound, int rightBound)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int max = rightBound - leftBound + 1;
    for (int idx = index; idx < max; idx+=stride) {

        energy[idx] = sobel[leftBound+idx];
        seamsTemporaries[idx*height] = leftBound+idx;
        int indexCurrent = leftBound+idx;
        for (int j = 1; j < height; j++) {
            indexCurrent = indexes[indexCurrent];
            energy[idx] += sobel[indexCurrent];
            seamsTemporaries[idx*height+j] = indexCurrent%width;
        }
    }
}
__global__ void calculateAllSeams2(int width, int height, int* indexes, float* sobel, int* seamsTemporaries, float* energy, int leftBound, int rightBound)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int max = rightBound - leftBound + 1;
    for (int idx = index; idx < max; idx+=stride) {

        energy[idx] = sobel[(height-1)*width+leftBound+idx];
        seamsTemporaries[idx*height + height-1] = leftBound+idx;
        int indexCurrent = (height-2)*width+leftBound+idx;
        for (int j = height-2; j >= 0; j--) {
            indexCurrent = indexes[indexCurrent];
            energy[idx] += sobel[indexCurrent];
            seamsTemporaries[idx*height+j] = indexCurrent%width;
        }
    }
}

__global__ void keepFirstSeam(int height, int* seamsTemporaries, float* energy, int* seam, int sizeWindow)
{
    int idx = 0;
    float energyMin = energy[0];
    for (int i = 1; i < sizeWindow; i++) {
        if (energy[i] < energyMin) {
            energyMin = energy[i];
            idx = i;
        }
    }

    memcpy(seam, &seamsTemporaries[idx*height], sizeof(int)*height);
}
__global__ void keepFirstSeamRight(int height, int* seamsTemporaries, float* energy, int* seam, int sizeWindow)
{
    int idx = 0;
    float energyMin = energy[0];
    for (int i = 1; i < sizeWindow; i++) {
        if (energy[i] <= energyMin) {
            energyMin = energy[i];
            idx = i;
        }
    }

    memcpy(seam, &seamsTemporaries[idx*height], sizeof(int)*height);
}

void seamResizeVideo(int width, int height, int widthDst, int sizeBuffer, unsigned char** dataIn, unsigned char** dataOut)
{   
    int blockSize = 1024;
    int widthHalf = width/2;
    int nbAdd = (widthDst-width)/2;
    int searchWindow = 460*widthHalf/1024;
    int leftBoundLeft = 0;
    int rightBoundLeft = searchWindow-1;
    int leftBoundRight = widthHalf - searchWindow;
    int rightBoundRight = widthHalf-1;

    // -- create streams
    hipStream_t streamLeft, streamRight;
    hipStreamCreate(&streamLeft);
    hipStreamCreate(&streamRight);

    // -- mean sobel for all frames
    float* sobelAllLeft;
    float* sobelAllRight;
    hipMalloc((void **)&sobelAllLeft, widthHalf*height*sizeof(float));
    hipMalloc((void **)&sobelAllRight, widthHalf*height*sizeof(float));
    hipMemset(sobelAllLeft, 0, widthHalf*height*sizeof(float));
    hipMemset(sobelAllRight, 0, widthHalf*height*sizeof(float));
    float* sobelAll2Left;
    float* sobelAll2Right;
    hipMalloc((void **)&sobelAll2Left, widthHalf*height*sizeof(float));
    hipMalloc((void **)&sobelAll2Right, widthHalf*height*sizeof(float));

    // -- max 120 frames at the time
    int nbFramesTreated = 0;
    while (nbFramesTreated < sizeBuffer) {

        int nbToTreat = 120;
        if (sizeBuffer - nbFramesTreated < 120) {
            nbToTreat = sizeBuffer - nbFramesTreated;
        }

        // -- copy datas to GPU
        unsigned char* dataRGB;
        hipMalloc(&dataRGB, width*height*3*nbToTreat*sizeof(unsigned char));
        for (int i = 0; i < nbToTreat; i++) {
            hipMemcpy(&dataRGB[i*width*height*3], dataIn[nbFramesTreated+i], width*height*3*sizeof(unsigned char), hipMemcpyHostToDevice);
        }

        // -- separate left and right
        unsigned char* dataRGBLeft;
        unsigned char* dataRGBRight;
        int numBlocksSplit = ((width*height*3*nbToTreat) + blockSize - 1) / blockSize;
        hipMalloc(&dataRGBLeft, widthHalf*height*3*nbToTreat*sizeof(unsigned char));
        hipMalloc(&dataRGBRight, widthHalf*height*3*nbToTreat*sizeof(unsigned char));
        cutFrame<<<numBlocksSplit, blockSize>>>(width, height, nbToTreat, dataRGB, dataRGBLeft, dataRGBRight);
        hipDeviceSynchronize();

        // -- calculate grayscale image
        float* grayImageLeft;
        float* grayImageRight;
        int numBlocksFilter = ((widthHalf*height*nbToTreat) + blockSize - 1) / blockSize;
        hipMalloc((void **)&grayImageLeft, widthHalf*height*nbToTreat*sizeof(float));
        hipMalloc((void **)&grayImageRight, widthHalf*height*nbToTreat*sizeof(float));
        grayscale<<<numBlocksFilter, blockSize, 0, streamLeft>>>(widthHalf, height, nbToTreat, dataRGBLeft, grayImageLeft);
        grayscale<<<numBlocksFilter, blockSize, 0, streamRight>>>(widthHalf, height, nbToTreat, dataRGBRight, grayImageRight);
        hipDeviceSynchronize();

        // -- calculate sobel
        float* sobelImageLeft;
        float* sobelImageRight;
        hipMalloc((void **)&sobelImageLeft, widthHalf*height*nbToTreat*sizeof(float));
        hipMalloc((void **)&sobelImageRight, widthHalf*height*nbToTreat*sizeof(float));
        // -- energy definition
        sobel<<<numBlocksFilter, blockSize, 0, streamLeft>>>(widthHalf, height, nbToTreat, grayImageLeft, sobelImageLeft);
        sobel<<<numBlocksFilter, blockSize, 0, streamRight>>>(widthHalf, height, nbToTreat, grayImageRight, sobelImageRight);
        hipDeviceSynchronize();

        // -- global energy definition
        int numBlocksMerge = ((widthHalf*height) + blockSize - 1) / blockSize;
        for (int i = 0; i < nbToTreat; i++) {
            mergeSobel<<<numBlocksMerge, blockSize, 0, streamLeft>>>(widthHalf, height, i, sizeBuffer, sobelImageLeft, sobelAllLeft);
            mergeSobel<<<numBlocksMerge, blockSize, 0, streamRight>>>(widthHalf, height, i, sizeBuffer, sobelImageRight, sobelAllRight);
            hipDeviceSynchronize();
        }

        // Free temporary sobel memory
        hipFree(dataRGB);
        hipFree(dataRGBLeft);
        hipFree(dataRGBRight);
        hipFree(grayImageLeft);
        hipFree(grayImageRight);
        hipFree(sobelImageLeft);
        hipFree(sobelImageRight);

        nbFramesTreated += nbToTreat;
    }

    // -- indexes map definition
    int* indexesLeft;
    int* indexesRight;
    hipMalloc((void**)&indexesLeft, height*widthHalf*sizeof(int));
    hipMalloc((void**)&indexesRight, height*widthHalf*sizeof(int));
    // -- seams definitions
    int* seamsLeft;
    int* seamsRight;
    hipMalloc((void**)&seamsLeft, height*nbAdd*sizeof(int));
    hipMalloc((void**)&seamsRight, height*nbAdd*sizeof(int));

    // -- for other calculation of seams
    int* seamsAllLeft;
    int* seamsAllRight;
    hipMalloc((void**)&seamsAllLeft, height*searchWindow*sizeof(int));
    hipMalloc((void**)&seamsAllRight, height*searchWindow*sizeof(int));
    float* energySeamsLeft;
    float* energySeamsRight;
    hipMalloc((void**)&energySeamsLeft, searchWindow*sizeof(float));
    hipMalloc((void**)&energySeamsRight, searchWindow*sizeof(float));

    // -- start simulation removing
    int widthTemp = widthHalf;
    for (int i = 0; i < nbAdd; i++) {

        float* sobelInLeft;
        float* sobelOutLeft;
        float* sobelInRight;
        float* sobelOutRight;

        // -- alternate between
        if (i%2 == 0) {
            sobelInLeft = sobelAllLeft;
            sobelOutLeft = sobelAll2Left;
            sobelInRight = sobelAllRight;
            sobelOutRight = sobelAll2Right;
        }
        else {
            sobelInLeft = sobelAll2Left;
            sobelOutLeft = sobelAllLeft;
            sobelInRight = sobelAll2Right;
            sobelOutRight = sobelAllRight;
        }

        // -- index map calculation
        hipMemset(indexesLeft, 0, height*widthTemp*sizeof(int));
        hipMemset(indexesRight, 0, height*widthTemp*sizeof(int));
        int numBlocksEnergy = ((widthTemp*(height-1)) + blockSize - 1) / blockSize;
        computeIndexMap<<<numBlocksEnergy, blockSize, 0, streamLeft>>>(widthTemp, height, sobelInLeft, indexesLeft);
        computeIndexMap<<<numBlocksEnergy, blockSize, 0, streamRight>>>(widthTemp, height, sobelInRight, indexesRight);
        hipDeviceSynchronize();

        int numBlocksSeams = ((rightBoundLeft - leftBoundLeft + 1) + blockSize - 1) / blockSize;
        calculateAllSeams<<<numBlocksSeams, blockSize, 0, streamLeft>>>(widthTemp, height, indexesLeft, sobelInLeft, seamsAllLeft, energySeamsLeft, leftBoundLeft, rightBoundLeft);
        calculateAllSeams<<<numBlocksSeams, blockSize, 0, streamRight>>>(widthTemp, height, indexesRight, sobelInRight, seamsAllRight, energySeamsRight, leftBoundRight, rightBoundRight);
        hipDeviceSynchronize();

        keepFirstSeam<<<1,1, 0, streamLeft>>>(height, seamsAllLeft, energySeamsLeft, &seamsLeft[i*height], rightBoundLeft - leftBoundLeft + 1);
        keepFirstSeamRight<<<1,1, 0, streamRight>>>(height, seamsAllRight, energySeamsRight, &seamsRight[i*height], rightBoundRight - leftBoundRight + 1);
        hipDeviceSynchronize();

        // -- compute new sobel
        widthTemp--;
        rightBoundLeft--;
        rightBoundRight--;

        // -- remove seam from sobel
        int numBlocksRemove = ((widthTemp*height) + blockSize - 1) / blockSize;
        removeSeamFromSobel<<<numBlocksRemove, blockSize, 0, streamLeft>>>(widthTemp, height, sobelInLeft, sobelOutLeft, &seamsLeft[i*height]);
        removeSeamFromSobel<<<numBlocksRemove, blockSize, 0, streamRight>>>(widthTemp, height, sobelInRight, sobelOutRight, &seamsRight[i*height]);
        hipDeviceSynchronize();
    }

    // -- repercute offset of previous seam to all next seams
    int numBlocksReper = (height + blockSize - 1) / blockSize;
    repercuteSeam<<<numBlocksReper, blockSize, 0, streamLeft>>>(nbAdd, seamsLeft, height);
    repercuteSeam<<<numBlocksReper, blockSize, 0, streamRight>>>(nbAdd, seamsRight, height);
    hipDeviceSynchronize();

    // Free temporary seam memory
    hipFree(sobelAllLeft);
    hipFree(sobelAllRight);
    hipFree(sobelAll2Left);
    hipFree(sobelAll2Right);
    hipFree(indexesLeft);
    hipFree(indexesRight);
    hipFree(seamsAllLeft);
    hipFree(seamsAllRight);
    hipFree(energySeamsLeft);
    hipFree(energySeamsRight);

    nbFramesTreated = 0;
    while (nbFramesTreated < sizeBuffer) {

        int nbToTreat = 120;
        if (sizeBuffer - nbFramesTreated < 120) {
            nbToTreat = sizeBuffer - nbFramesTreated;
        }

        // -- copy datas to GPU
        unsigned char* dataRGB;
        hipMalloc(&dataRGB, width*height*3*nbToTreat*sizeof(unsigned char));
        for (int i = 0; i < nbToTreat; i++) {
            hipMemcpy(&dataRGB[i*width*height*3], dataIn[nbFramesTreated+i], width*height*3*sizeof(unsigned char), hipMemcpyHostToDevice);
        }

        // -- separate left and right
        unsigned char* dataRGBLeft;
        unsigned char* dataRGBRight;
        int numBlocksSplit = ((width*height*3*nbToTreat) + blockSize - 1) / blockSize;
        hipMalloc(&dataRGBLeft, widthHalf*height*3*nbToTreat*sizeof(unsigned char));
        hipMalloc(&dataRGBRight, widthHalf*height*3*nbToTreat*sizeof(unsigned char));
        cutFrame<<<numBlocksSplit, blockSize>>>(width, height, nbToTreat, dataRGB, dataRGBLeft, dataRGBRight);
        hipDeviceSynchronize();

        // -- add new seams in order
        unsigned char* dataOutDeviceLeft;
        unsigned char* dataOutDevice2Left;
        unsigned char* dataOutDeviceRight;
        unsigned char* dataOutDevice2Right;
        hipMalloc(&dataOutDeviceLeft, widthDst/2*height*3*nbToTreat*sizeof(unsigned char));
        hipMalloc(&dataOutDeviceRight, widthDst/2*height*3*nbToTreat*sizeof(unsigned char));
        hipMalloc(&dataOutDevice2Left, widthDst/2*height*3*nbToTreat*sizeof(unsigned char));
        hipMalloc(&dataOutDevice2Right, widthDst/2*height*3*nbToTreat*sizeof(unsigned char));
        hipMemcpy(dataOutDeviceLeft, dataRGBLeft, widthHalf*height*3*nbToTreat*sizeof(unsigned char), hipMemcpyDeviceToDevice);
        hipMemcpy(dataOutDeviceRight, dataRGBRight, widthHalf*height*3*nbToTreat*sizeof(unsigned char), hipMemcpyDeviceToDevice);

        widthTemp = widthHalf+1;
        for (int i = 0; i < nbAdd; i++) {
            int numBlocksAdd = ((widthTemp*height*3*nbToTreat) + blockSize - 1) / blockSize;
            if (i%2 == 0) {
                addSeamToImage<<<numBlocksAdd, blockSize, 0, streamLeft>>>(widthTemp, height, nbToTreat, dataOutDeviceLeft, dataOutDevice2Left, &seamsLeft[i*height]);
                addSeamToImage<<<numBlocksAdd, blockSize, 0, streamRight>>>(widthTemp, height, nbToTreat, dataOutDeviceRight, dataOutDevice2Right, &seamsRight[i*height]);
            }
            else {
                addSeamToImage<<<numBlocksAdd, blockSize, 0, streamLeft>>>(widthTemp, height, nbToTreat, dataOutDevice2Left, dataOutDeviceLeft, &seamsLeft[i*height]);
                addSeamToImage<<<numBlocksAdd, blockSize, 0, streamRight>>>(widthTemp, height, nbToTreat, dataOutDevice2Right, dataOutDeviceRight, &seamsRight[i*height]);
            }
            hipDeviceSynchronize();
            widthTemp++;
        }

        unsigned char* dataOutDevice;
        hipMalloc((void **)&dataOutDevice, widthDst*height*3*nbToTreat*sizeof(unsigned char));
        int numBlocksCopy = ((widthDst*height*3*nbToTreat) + blockSize - 1) / blockSize;
        if (nbAdd%2 == 0) {copyImage<<<numBlocksCopy,blockSize>>>(widthDst, height, nbToTreat, dataOutDeviceLeft, dataOutDeviceRight, dataOutDevice);}
        else {copyImage<<<numBlocksCopy,blockSize>>>(widthDst, height, nbToTreat, dataOutDevice2Left, dataOutDevice2Right, dataOutDevice);}
        hipDeviceSynchronize();

        // -- copy final image
        for (int i = 0; i < nbToTreat; i++) {
            hipMemcpy(dataOut[nbFramesTreated+i], &dataOutDevice[i*widthDst*height*3], widthDst*height*3*sizeof(unsigned char), hipMemcpyDeviceToHost);
        }

        // Free final images
        hipFree(dataRGB);
        hipFree(dataRGBLeft);
        hipFree(dataRGBRight);
        hipFree(dataOutDevice);
        hipFree(dataOutDeviceLeft);
        hipFree(dataOutDeviceRight);
        hipFree(dataOutDevice2Left);
        hipFree(dataOutDevice2Right);

        nbFramesTreated += nbToTreat;
    }

    // Free global memory
    hipFree(seamsLeft);
    hipFree(seamsRight);
    hipStreamDestroy(streamLeft);
    hipStreamDestroy(streamRight);
}