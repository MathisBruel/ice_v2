#include "hip/hip_runtime.h"
#include <seam.cuh>

__global__ void cutFrame(int width, int height, unsigned char* dataRGB, unsigned char* left, unsigned char* right)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int idx = index; idx < width*height*3; idx+=stride) {

        int comp = idx%3;
        int idxTemp = (idx-comp)/3;
        int i = idxTemp%width;
        int j = (idxTemp-i)/width;
        int tempWidth = width/2;

        if (i < tempWidth) {
            left[(j*tempWidth+i)*3+comp] = dataRGB[idx];
        }
        else {
            right[((j*tempWidth)+i-tempWidth)*3+comp] = dataRGB[idx];
        }
    }
}

__global__ void grayscale(int width, int height, unsigned char* dataRGB, float* dataOutGray)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < width*height; i+=stride) {
        dataOutGray[i] = (float)(dataRGB[i*3])*0.2989f + (float)(dataRGB[i*3+1])*0.587f + (float)(dataRGB[i*3+2])*0.114f;
        dataOutGray[i] /= 255;
    }
}

__global__ void sobel(int width, int height, float* dataInGray, float* dataOutSobel, float* dataOutSobel2)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int idx = index; idx < height*width; idx+=stride) {

        int i = idx%width;
        int j = (idx - i)/width;
        float color1 = (i == 0 || j == 0) ? 0 : dataInGray[idx - width - 1];
        float color2 = (j == 0) ? 0 : dataInGray[idx - width];
        float color3 = (i == width-1 || j == 0) ? 0 :  dataInGray[idx - width + 1];
        float color4 = (i == 0) ? 0 : dataInGray[idx - 1];
        float color5 = (i == width-1) ? 0 : dataInGray[idx + 1];
        float color6 = (i == 0 || j == height-1) ? 0 : dataInGray[idx + width - 1];
        float color7 = (j == height-1) ? 0 : dataInGray[idx + width];
        float color8 = (i == width-1 || j == height-1) ? 0 : dataInGray[idx + width + 1];

        float dx = color3 + 2*color5 + color8 - color1 - 2*color4 - color6;
        float dy = color6 + 2*color7 + color8 - color1 - 2*color2 - color3;
        dataOutSobel[idx] = dx*dx + dy*dy;
        dataOutSobel2[idx] = dataOutSobel[idx];
    }
}
__global__ void sobelSimple(int width, int height, float* dataInGray, float* dataOutSobel)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int idx = index; idx < height*width; idx+=stride) {

        int i = idx%width;
        int j = (idx - i)/width;
        float color1 = (i == 0 || j == 0) ? 0 : dataInGray[idx - width - 1];
        float color2 = (j == 0) ? 0 : dataInGray[idx - width];
        float color3 = (i == width-1 || j == 0) ? 0 :  dataInGray[idx - width + 1];
        float color4 = (i == 0) ? 0 : dataInGray[idx - 1];
        float color5 = (i == width-1) ? 0 : dataInGray[idx + 1];
        float color6 = (i == 0 || j == height-1) ? 0 : dataInGray[idx + width - 1];
        float color7 = (j == height-1) ? 0 : dataInGray[idx + width];
        float color8 = (i == width-1 || j == height-1) ? 0 : dataInGray[idx + width + 1];

        float dx = color3 + 2*color5 + color8 - color1 - 2*color4 - color6;
        float dy = color6 + 2*color7 + color8 - color1 - 2*color2 - color3;
        dataOutSobel[idx] = dx*dx + dy*dy;
    }
}
__global__ void sobelComplex(int width, int height, float* dataInGray, float* dataInSobel, float* dataOutSobel)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int idx = index; idx < height*width; idx+=stride) {

        int i = idx%width;
        int j = (idx - i)/width;
        float color1 = (i == 0 || j == 0) ? 0 : dataInGray[idx - width - 1];
        float color2 = (j == 0) ? 0 : dataInGray[idx - width];
        float color3 = (i == width-1 || j == 0) ? 0 :  dataInGray[idx - width + 1];
        float color4 = (i == 0) ? 0 : dataInGray[idx - 1];
        float color5 = (i == width-1) ? 0 : dataInGray[idx + 1];
        float color6 = (i == 0 || j == height-1) ? 0 : dataInGray[idx + width - 1];
        float color7 = (j == height-1) ? 0 : dataInGray[idx + width];
        float color8 = (i == width-1 || j == height-1) ? 0 : dataInGray[idx + width + 1];

        float dx = color3 + 2*color5 + color8 - color1 - 2*color4 - color6;
        float dy = color6 + 2*color7 + color8 - color1 - 2*color2 - color3;
        float energyNew = (dx*dx + dy*dy);
        dataOutSobel[idx] = dataInSobel[idx] + (energyNew - dataInSobel[idx])*0.3;
        //dataInSobel[idx] = energyNew;
    }
}

__global__ void removeSeamFromSobel(int width, int height, float *dataSobel, float *dataSobelNew, int* seam)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int idx = index; idx < width*height; idx+=stride) {

        int i = idx%width;
        int j = (idx-i)/width;
        int idx2 = j*(width+1)+i;

        if (i < seam[j]) {dataSobelNew[idx] = dataSobel[idx2];}
        else {dataSobelNew[idx] = dataSobel[idx2+1];}
    }
}

__global__ void addSeamToImage(int width, int height, unsigned char* dataIn, unsigned char* dataOut, int* seam)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int idx = index; idx < width*height*3; idx+=stride) {

        int comp = idx%3;
        int idxTemp = (idx-comp)/3;
        int i = idxTemp%width;
        int j = (idxTemp-i)/width;
        int idx2 = (j*(width-1)+i)*3+comp;

        // -- before seam
        if (i <= seam[j]) {dataOut[idx] = dataIn[idx2];}
        // -- after seam
        else if (i > seam[j] + 1) {dataOut[idx] = dataIn[idx2-3];}
        // -- duplicate seam
        else {
            int value = dataIn[idx2-3];
            if (seam[j]+1 < width-1) {
                value += dataIn[idx2];
            }
            value /= 2;
            dataOut[idx] = (unsigned char)value;
        }
    }
}

__global__ void copyImage(int width, int height, unsigned char* dataInLeft, unsigned char* dataInRight, unsigned char* dataOut) 
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int idx = index; idx < width*height*3; idx+=stride) {
        
        unsigned char comp = idx%3;
        int idxTemp = (idx-comp)/3;
        short i = idxTemp%width;
        short j = (idxTemp-i)/width;
        int tempWidth = width/2;

        if (i < tempWidth) {
            dataOut[idx] = dataInLeft[((j*tempWidth)+i)*3+comp];
        }
        else {
            dataOut[idx] = dataInRight[((j*tempWidth)+i-tempWidth)*3+comp];
        }
    }
}

__global__ void computeIndexMap(int width, int height, float* sobelImage, int* indexes)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int idx = index; idx < width*(height-1); idx+=stride) {

        int i = idx%width;
        int tempIdx = idx+width;
        if (i == 0) {
            float middle = sobelImage[tempIdx];
            float right = sobelImage[tempIdx+1];

            if (middle <= right) {indexes[idx] = tempIdx;}
            else {indexes[idx] = tempIdx+1;}
        }
        else if (i == width-1) {
            float left = sobelImage[tempIdx-1];
            float middle = sobelImage[tempIdx];

            if (middle <= left) {indexes[idx] = tempIdx;}
            else {indexes[idx] = tempIdx-1;}
        }
        else {
            float left = sobelImage[tempIdx-1];
            float middle = sobelImage[tempIdx];
            float right = sobelImage[tempIdx+1];

            if (middle <= left && middle <= right) {indexes[idx] = tempIdx;}
            else if (left <= right) {indexes[idx] = tempIdx-1;}
            else {indexes[idx] = tempIdx+1;}
        }
    }
}

__global__ void computeIndexMap2(int width, int height, float* sobelImage, int* indexes)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int idx = index; idx < width*(height-1); idx+=stride) {

        int i = idx%width;
        int tempIdx = idx;
        if (i == 0) {
            float middle = sobelImage[tempIdx];
            float right = sobelImage[tempIdx+1];

            if (middle <= right) {indexes[idx] = tempIdx;}
            else {indexes[idx] = tempIdx+1;}
        }
        else if (i == width-1) {
            float left = sobelImage[tempIdx-1];
            float middle = sobelImage[tempIdx];

            if (middle <= left) {indexes[idx] = tempIdx;}
            else {indexes[idx] = tempIdx-1;}
        }
        else {
            float left = sobelImage[tempIdx-1];
            float middle = sobelImage[tempIdx];
            float right = sobelImage[tempIdx+1];

            if (middle <= left && middle <= right) {indexes[idx] = tempIdx;}
            else if (left <= right) {indexes[idx] = tempIdx-1;}
            else {indexes[idx] = tempIdx+1;}
        }
    }
}

__global__ void repercuteSeam(int nbSeamsImpact, int* seams, int height)
{
    for (int i = 0; i < nbSeamsImpact; i++) {
        for (int j = i+1; j < nbSeamsImpact; j++) {

            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            for (int idx = index; idx < height; idx+=stride) {
        
                if (seams[j*height+idx] >= seams[i*height+idx]) {
                    seams[j*height+idx] += 2;
                }
            }
        }
    }
}

__global__ void calculateAllSeams(int width, int height, int* indexes, float* sobel, int* seamsTemporaries, float* energy, int leftBound, int rightBound)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int max = rightBound - leftBound + 1;
    for (int idx = index; idx < max; idx+=stride) {

        energy[idx] = sobel[leftBound+idx];
        seamsTemporaries[idx*height] = leftBound+idx;
        int indexCurrent = leftBound+idx;
        for (int j = 1; j < height; j++) {
            indexCurrent = indexes[indexCurrent];
            energy[idx] += sobel[indexCurrent];
            seamsTemporaries[idx*height+j] = indexCurrent%width;
        }
    }
}

__global__ void calculateAllSeams2(int width, int height, int* indexes, float* sobel, int* seamsTemporaries, float* energy, int leftBound, int rightBound)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int max = rightBound - leftBound + 1;
    for (int idx = index; idx < max; idx+=stride) {

        energy[idx] = sobel[(height-1)*width+leftBound+idx];
        seamsTemporaries[idx*height + height-1] = leftBound+idx;
        int indexCurrent = (height-2)*width+leftBound+idx;
        for (int j = height-2; j >= 0; j--) {
            indexCurrent = indexes[indexCurrent];
            energy[idx] += sobel[indexCurrent];
            seamsTemporaries[idx*height+j] = indexCurrent%width;
        }
    }
}

__global__ void keepFirstSeam(int height, int* seamsTemporaries, float* energy, int* seam, int sizeWindow, float* energyMax)
{
    int idx = 0;
    float energyMin = energy[0];
    for (int i = 1; i < sizeWindow; i++) {
        if (energy[i] < energyMin) {
            energyMin = energy[i];
            idx = i;
        }
    }

    energyMax[0] += energy[idx];

    for (int i = 0; i < height; i++) {
        seam[i] = seamsTemporaries[idx*height+i];
    }
}
__global__ void keepFirstSeamRight(int height, int* seamsTemporaries, float* energy, int* seam, int sizeWindow, float* energyMax)
{
    int idx = 0;
    float energyMin = energy[0];
    for (int i = 1; i < sizeWindow; i++) {
        if (energy[i] <= energyMin) {
            energyMin = energy[i];
            idx = i;
        }
    }

    energyMax[0] += energy[idx];

    for (int i = 0; i < height; i++) {
        seam[i] = seamsTemporaries[idx*height+i];
    }
}

__global__ void calculateDistanceToOldSeam(int height, int* seamPrevious, int* seamsTemporaries, float* energy, int* distance, int sizeWindow)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int idx = index; idx < sizeWindow; idx+=stride) {

        distance[idx] = 0;
        for (int j = 0; j < height; j++) {
            distance[idx] += abs(seamPrevious[j] - seamsTemporaries[idx*height+j]);
        }
        distance[idx] *= energy[idx];
    }
}

__global__ void keepFirstMotionSeamLeft(int height, int* seamsTemporaries, float* energy, int* seam, int sizeWindow)
{
    int idx = 0;
    float energyMin = energy[0];
    for (int i = 1; i < sizeWindow; i++) {
        if (energy[i] < energyMin) {
            energyMin = energy[i];
            idx = i;
        }
    }

    memcpy(seam, &seamsTemporaries[idx*height], sizeof(int)*height);
}
__global__ void keepFirstMotionSeamRight(int height, int* seamsTemporaries, float* energy, int* seam, int sizeWindow)
{
    int idx = 0;
    float energyMin = energy[0];
    for (int i = 1; i < sizeWindow; i++) {
        if (energy[i] <= energyMin) {
            energyMin = energy[i];
            idx = i;
        }
    }

    memcpy(seam, &seamsTemporaries[idx*height], sizeof(int)*height);
}

__global__ void keepCloserSeam(int height, int* seamsTemporaries, int* distance, int* seam, int sizeWindow)
{
    int idx = 0;
    int minDistance = distance[0];
    for (int i = 1; i < sizeWindow; i++) {
        if (distance[i] < minDistance) {
            minDistance = distance[i];
            idx = i;
        }
    }

    memcpy(seam, &seamsTemporaries[idx*height], sizeof(int)*height);
}

void addResizeVideoFirst(int width, int height, int widthDst, unsigned char* dataIn, unsigned char* dataOut, int* seamsOutLeft, int* seamsOutRight)
{   
    int blockSize = 1024;
    int widthHalf = width/2;
    int nbAdd = (widthDst-width)/2;
    int searchWindow = 460*widthHalf/1024;
    int leftBoundLeft = 0;
    int rightBoundLeft = searchWindow-1;
    int leftBoundRight = widthHalf - searchWindow;
    int rightBoundRight = widthHalf-1;

    // -- create streams
    hipStream_t streamLeft, streamRight;
    hipStreamCreate(&streamLeft);
    hipStreamCreate(&streamRight);
    hipStream_t streamLeft2, streamRight2;
    hipStreamCreate(&streamLeft2);
    hipStreamCreate(&streamRight2);

    // -- copy datas to GPU
    unsigned char* dataRGB;
    hipMalloc(&dataRGB, width*height*3*sizeof(unsigned char));
    hipMemcpy(dataRGB, dataIn, width*height*3*sizeof(unsigned char), hipMemcpyHostToDevice);

    // -- separate left and right
    unsigned char* dataRGBLeft;
    unsigned char* dataRGBRight;
    int numBlocksSplit = ((width*height*3) + blockSize - 1) / blockSize;
    hipMalloc(&dataRGBLeft, widthHalf*height*3*sizeof(unsigned char));
    hipMalloc(&dataRGBRight, widthHalf*height*3*sizeof(unsigned char));
    cutFrame<<<numBlocksSplit, blockSize>>>(width, height, dataRGB, dataRGBLeft, dataRGBRight);
    hipDeviceSynchronize();

    // -- calculate grayscale image
    float* grayImageLeft;
    float* grayImageRight;
    int numBlocksFilter = ((widthHalf*height) + blockSize - 1) / blockSize;
    hipMalloc((void **)&grayImageLeft, widthHalf*height*sizeof(float));
    hipMalloc((void **)&grayImageRight, widthHalf*height*sizeof(float));
    grayscale<<<numBlocksFilter, blockSize, 0, streamLeft>>>(widthHalf, height, dataRGBLeft, grayImageLeft);
    grayscale<<<numBlocksFilter, blockSize, 0, streamRight>>>(widthHalf, height, dataRGBRight, grayImageRight);
    hipDeviceSynchronize();

    // -- calculate sobel
    float* sobelImageLeftUp;
    float* sobelImageRightUp;
    float* sobelImageLeftDown;
    float* sobelImageRightDown;
    hipMalloc((void **)&sobelImageLeftUp, widthHalf*height*sizeof(float));
    hipMalloc((void **)&sobelImageRightUp, widthHalf*height*sizeof(float));
    hipMalloc((void **)&sobelImageLeftDown, widthHalf*height*sizeof(float));
    hipMalloc((void **)&sobelImageRightDown, widthHalf*height*sizeof(float));
    float* sobelImage2LeftUp;
    float* sobelImage2RightUp;
    float* sobelImage2LeftDown;
    float* sobelImage2RightDown;
    hipMalloc((void **)&sobelImage2LeftUp, widthHalf*height*sizeof(float));
    hipMalloc((void **)&sobelImage2RightUp, widthHalf*height*sizeof(float));
    hipMalloc((void **)&sobelImage2LeftDown, widthHalf*height*sizeof(float));
    hipMalloc((void **)&sobelImage2RightDown, widthHalf*height*sizeof(float));
    // -- energy definition
    sobel<<<numBlocksFilter, blockSize, 0, streamLeft>>>(widthHalf, height, grayImageLeft, sobelImageLeftUp, sobelImageLeftDown);
    sobel<<<numBlocksFilter, blockSize, 0, streamRight>>>(widthHalf, height, grayImageRight, sobelImageRightUp, sobelImageRightDown);
    hipDeviceSynchronize();

    // -- indexes map definition
    int* indexesLeftUp;
    int* indexesRightUp;
    hipMalloc((void**)&indexesLeftUp, height*widthHalf*sizeof(int));
    hipMalloc((void**)&indexesRightUp, height*widthHalf*sizeof(int));
    int* indexesLeftDown;
    int* indexesRightDown;
    hipMalloc((void**)&indexesLeftDown, height*widthHalf*sizeof(int));
    hipMalloc((void**)&indexesRightDown, height*widthHalf*sizeof(int));
    // -- seams definitions
    int* seamsLeftUp;
    int* seamsRightUp;
    hipMalloc((void**)&seamsLeftUp, height*nbAdd*sizeof(int));
    hipMalloc((void**)&seamsRightUp, height*nbAdd*sizeof(int));
    int* seamsLeftDown;
    int* seamsRightDown;
    hipMalloc((void**)&seamsLeftDown, height*nbAdd*sizeof(int));
    hipMalloc((void**)&seamsRightDown, height*nbAdd*sizeof(int));

    // -- for other calculation of seams
    int* seamsAllLeftUp;
    int* seamsAllRightUp;
    hipMalloc((void**)&seamsAllLeftUp, height*searchWindow*sizeof(int));
    hipMalloc((void**)&seamsAllRightUp, height*searchWindow*sizeof(int));
    int* seamsAllLeftDown;
    int* seamsAllRightDown;
    hipMalloc((void**)&seamsAllLeftDown, height*searchWindow*sizeof(int));
    hipMalloc((void**)&seamsAllRightDown, height*searchWindow*sizeof(int));
    float* energySeamsLeftUp;
    float* energySeamsRightUp;
    hipMalloc((void**)&energySeamsLeftUp, searchWindow*sizeof(float));
    hipMalloc((void**)&energySeamsRightUp, searchWindow*sizeof(float));
    float* energySeamsLeftDown;
    float* energySeamsRightDown;
    hipMalloc((void**)&energySeamsLeftDown, searchWindow*sizeof(float));
    hipMalloc((void**)&energySeamsRightDown, searchWindow*sizeof(float));
    float* energySumUpLeft;
    float* energySumUpRight;
    float* energySumDownLeft;
    float* energySumDownRight;
    hipMalloc((void**)&energySumUpLeft, sizeof(float));
    hipMalloc((void**)&energySumUpRight, sizeof(float));
    hipMalloc((void**)&energySumDownLeft, sizeof(float));
    hipMalloc((void**)&energySumDownRight, sizeof(float));
    hipMemset(energySumUpLeft, 0, sizeof(float));
    hipMemset(energySumUpRight, 0, sizeof(float));
    hipMemset(energySumDownLeft, 0, sizeof(float));
    hipMemset(energySumDownRight, 0, sizeof(float));

    // -- start simulation removing
    int widthTemp = widthHalf;
    for (int i = 0; i < nbAdd; i++) {

        float* sobelInLeftUp;
        float* sobelOutLeftUp;
        float* sobelInRightUp;
        float* sobelOutRightUp;
        float* sobelInLeftDown;
        float* sobelOutLeftDown;
        float* sobelInRightDown;
        float* sobelOutRightDown;

        // -- alternate between
        if (i%2 == 0) {
            sobelInLeftUp = sobelImageLeftUp;
            sobelOutLeftUp = sobelImage2LeftUp;
            sobelInRightUp = sobelImageRightUp;
            sobelOutRightUp = sobelImage2RightUp;
            sobelInLeftDown = sobelImageLeftDown;
            sobelOutLeftDown = sobelImage2LeftDown;
            sobelInRightDown = sobelImageRightDown;
            sobelOutRightDown = sobelImage2RightDown;
        }
        else {
            sobelInLeftUp = sobelImage2LeftUp;
            sobelOutLeftUp = sobelImageLeftUp;
            sobelInRightUp = sobelImage2RightUp;
            sobelOutRightUp = sobelImageRightUp;
            sobelInLeftDown = sobelImage2LeftDown;
            sobelOutLeftDown = sobelImageLeftDown;
            sobelInRightDown = sobelImage2RightDown;
            sobelOutRightDown = sobelImageRightDown;
        }

        // -- index map calculation
        hipMemset(indexesLeftUp, 0, height*widthTemp*sizeof(int));
        hipMemset(indexesRightUp, 0, height*widthTemp*sizeof(int));
        hipMemset(indexesLeftDown, 0, height*widthTemp*sizeof(int));
        hipMemset(indexesRightDown, 0, height*widthTemp*sizeof(int));
        int numBlocksEnergy = ((widthTemp*(height-1)) + blockSize - 1) / blockSize;
        computeIndexMap<<<numBlocksEnergy, blockSize, 0, streamLeft>>>(widthTemp, height, sobelInLeftUp, indexesLeftUp);
        computeIndexMap<<<numBlocksEnergy, blockSize, 0, streamRight>>>(widthTemp, height, sobelInRightUp, indexesRightUp);
        computeIndexMap2<<<numBlocksEnergy, blockSize, 0, streamLeft2>>>(widthTemp, height, sobelInLeftDown, indexesLeftDown);
        computeIndexMap2<<<numBlocksEnergy, blockSize, 0, streamRight2>>>(widthTemp, height, sobelInRightDown, indexesRightDown);
        hipDeviceSynchronize();

        int numBlocksSeams = ((rightBoundLeft - leftBoundLeft + 1) + blockSize - 1) / blockSize;
        calculateAllSeams<<<numBlocksSeams, blockSize, 0, streamLeft>>>(widthTemp, height, indexesLeftUp, sobelInLeftUp, seamsAllLeftUp, energySeamsLeftUp, leftBoundLeft, rightBoundLeft);
        calculateAllSeams<<<numBlocksSeams, blockSize, 0, streamRight>>>(widthTemp, height, indexesRightUp, sobelInRightUp, seamsAllRightUp, energySeamsRightUp, leftBoundRight, rightBoundRight);
        calculateAllSeams2<<<numBlocksSeams, blockSize, 0, streamLeft2>>>(widthTemp, height, indexesLeftDown, sobelInLeftDown, seamsAllLeftDown, energySeamsLeftDown, leftBoundLeft, rightBoundLeft);
        calculateAllSeams2<<<numBlocksSeams, blockSize, 0, streamRight2>>>(widthTemp, height, indexesRightDown, sobelInRightDown, seamsAllRightDown, energySeamsRightDown, leftBoundRight, rightBoundRight);
        hipDeviceSynchronize();

        keepFirstSeam<<<1,1, 0, streamLeft>>>(height, seamsAllLeftUp, energySeamsLeftUp, &seamsLeftUp[i*height], rightBoundLeft - leftBoundLeft + 1, energySumUpLeft);
        keepFirstSeamRight<<<1,1, 0, streamRight>>>(height, seamsAllRightUp, energySeamsRightUp, &seamsRightUp[i*height], rightBoundRight - leftBoundRight + 1, energySumUpRight);
        keepFirstSeam<<<1,1, 0, streamLeft2>>>(height, seamsAllLeftDown, energySeamsLeftDown, &seamsLeftDown[i*height], rightBoundLeft - leftBoundLeft + 1, energySumDownLeft);
        keepFirstSeamRight<<<1,1, 0, streamRight2>>>(height, seamsAllRightDown, energySeamsRightDown, &seamsRightDown[i*height], rightBoundRight - leftBoundRight + 1, energySumDownRight);
        hipDeviceSynchronize();

        // -- compute new sobel
        widthTemp--;
        rightBoundLeft--;
        rightBoundRight--;

        // -- remove seam from sobel
        int numBlocksRemove = ((widthTemp*height) + blockSize - 1) / blockSize;
        removeSeamFromSobel<<<numBlocksRemove, blockSize, 0, streamLeft>>>(widthTemp, height, sobelInLeftUp, sobelOutLeftUp, &seamsLeftUp[i*height]);
        removeSeamFromSobel<<<numBlocksRemove, blockSize, 0, streamRight>>>(widthTemp, height, sobelInRightUp, sobelOutRightUp, &seamsRightUp[i*height]);
        removeSeamFromSobel<<<numBlocksRemove, blockSize, 0, streamLeft2>>>(widthTemp, height, sobelInLeftDown, sobelOutLeftDown, &seamsLeftDown[i*height]);
        removeSeamFromSobel<<<numBlocksRemove, blockSize, 0, streamRight2>>>(widthTemp, height, sobelInRightDown, sobelOutRightDown, &seamsRightDown[i*height]);
        hipDeviceSynchronize();
    }

    float energyUpLeft, energyUpRight, energyDownLeft, energyDownRight;
    hipMemcpy(&energyUpLeft, energySumUpLeft, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&energyUpRight, energySumUpRight, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&energyDownLeft, energySumDownLeft, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&energyDownRight, energySumDownRight, sizeof(float), hipMemcpyDeviceToHost);

    int* seamsLeft = energyUpLeft < energyDownLeft ? seamsLeftUp : seamsLeftDown;
    int* seamsRight = energyUpRight < energyDownRight ? seamsRightUp : seamsRightDown;

    hipMemcpy(seamsOutLeft, seamsLeft, height*nbAdd*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(seamsOutRight, seamsRight, height*nbAdd*sizeof(int), hipMemcpyDeviceToHost);

    // -- repercute offset of previous seam to all next seams
    int numBlocksReper = (height + blockSize - 1) / blockSize;
    repercuteSeam<<<numBlocksReper, blockSize, 0, streamLeft>>>(nbAdd, seamsLeft, height);
    repercuteSeam<<<numBlocksReper, blockSize, 0, streamRight>>>(nbAdd, seamsRight, height);
    hipDeviceSynchronize();

    // Free memory
    hipFree(grayImageLeft);
    hipFree(grayImageRight);
    hipFree(sobelImageLeftUp);
    hipFree(sobelImageRightUp);
    hipFree(sobelImage2LeftUp);
    hipFree(sobelImage2RightUp);
    hipFree(sobelImageLeftDown);
    hipFree(sobelImageRightDown);
    hipFree(sobelImage2LeftDown);
    hipFree(sobelImage2RightDown);
    hipFree(indexesLeftUp);
    hipFree(indexesRightUp);
    hipFree(seamsAllLeftUp);
    hipFree(seamsAllRightUp);
    hipFree(energySeamsLeftUp);
    hipFree(energySeamsRightUp);
    hipFree(indexesLeftDown);
    hipFree(indexesRightDown);
    hipFree(seamsAllLeftDown);
    hipFree(seamsAllRightDown);
    hipFree(energySeamsLeftDown);
    hipFree(energySeamsRightDown);
    hipFree(energySumUpLeft);
    hipFree(energySumDownLeft);
    hipFree(energySumUpRight);
    hipFree(energySumDownRight);

    // -- add new seams in order
    unsigned char* dataOutDeviceLeft;
    unsigned char* dataOutDevice2Left;
    unsigned char* dataOutDeviceRight;
    unsigned char* dataOutDevice2Right;
    hipMalloc(&dataOutDeviceLeft, widthDst/2*height*3*sizeof(unsigned char));
    hipMalloc(&dataOutDeviceRight, widthDst/2*height*3*sizeof(unsigned char));
    hipMalloc(&dataOutDevice2Left, widthDst/2*height*3*sizeof(unsigned char));
    hipMalloc(&dataOutDevice2Right, widthDst/2*height*3*sizeof(unsigned char));
    hipMemcpy(dataOutDeviceLeft, dataRGBLeft, widthHalf*height*3*sizeof(unsigned char), hipMemcpyDeviceToDevice);
    hipMemcpy(dataOutDeviceRight, dataRGBRight, widthHalf*height*3*sizeof(unsigned char), hipMemcpyDeviceToDevice);

    widthTemp = widthHalf+1;
    for (int i = 0; i < nbAdd; i++) {
        int numBlocksAdd = ((widthTemp*height*3) + blockSize - 1) / blockSize;
        if (i%2 == 0) {
            addSeamToImage<<<numBlocksAdd, blockSize, 0, streamLeft>>>(widthTemp, height, dataOutDeviceLeft, dataOutDevice2Left, &seamsLeft[i*height]);
            addSeamToImage<<<numBlocksAdd, blockSize, 0, streamRight>>>(widthTemp, height, dataOutDeviceRight, dataOutDevice2Right, &seamsRight[i*height]);
        }
        else {
            addSeamToImage<<<numBlocksAdd, blockSize, 0, streamLeft>>>(widthTemp, height, dataOutDevice2Left, dataOutDeviceLeft, &seamsLeft[i*height]);
            addSeamToImage<<<numBlocksAdd, blockSize, 0, streamRight>>>(widthTemp, height, dataOutDevice2Right, dataOutDeviceRight, &seamsRight[i*height]);
        }
        hipDeviceSynchronize();
        widthTemp++;
    }

    unsigned char* dataOutDevice;
    hipMalloc((void **)&dataOutDevice, widthDst*height*3*sizeof(unsigned char));
    int numBlocksCopy = ((widthDst*height*3) + blockSize - 1) / blockSize;
    if (nbAdd%2 == 0) {copyImage<<<numBlocksCopy,blockSize>>>(widthDst, height, dataOutDeviceLeft, dataOutDeviceRight, dataOutDevice);}
    else {copyImage<<<numBlocksCopy,blockSize>>>(widthDst, height, dataOutDevice2Left, dataOutDevice2Right, dataOutDevice);}
    hipDeviceSynchronize();

    // -- copy final image
    hipMemcpy(dataOut, dataOutDevice, widthDst*height*3*sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipFree(dataRGB);
    hipFree(dataRGBLeft);
    hipFree(dataRGBRight);
    hipFree(dataOutDevice);
    hipFree(dataOutDeviceLeft);
    hipFree(dataOutDeviceRight);
    hipFree(dataOutDevice2Left);
    hipFree(dataOutDevice2Right);
    hipFree(seamsLeftUp);
    hipFree(seamsLeftDown);
    hipFree(seamsRightUp);
    hipFree(seamsRightDown);
    hipStreamDestroy(streamLeft);
    hipStreamDestroy(streamRight);
    hipStreamDestroy(streamLeft2);
    hipStreamDestroy(streamRight2);
}
void addResizeVideoOther(int width, int height, int widthDst, unsigned char* dataIn, unsigned char* dataOut, int* seamsInLeft, int* seamsInRight)
{   
    int blockSize = 1024;
    int widthHalf = width/2;
    int nbAdd = (widthDst-width)/2;

    // -- create streams
    hipStream_t streamLeft, streamRight;
    hipStreamCreate(&streamLeft);
    hipStreamCreate(&streamRight);

    // -- copy datas to GPU
    unsigned char* dataRGB;
    hipMalloc(&dataRGB, width*height*3*sizeof(unsigned char));
    hipMemcpy(dataRGB, dataIn, width*height*3*sizeof(unsigned char), hipMemcpyHostToDevice);

    // -- separate left and right
    unsigned char* dataRGBLeft;
    unsigned char* dataRGBRight;
    int numBlocksSplit = ((width*height*3) + blockSize - 1) / blockSize;
    hipMalloc(&dataRGBLeft, widthHalf*height*3*sizeof(unsigned char));
    hipMalloc(&dataRGBRight, widthHalf*height*3*sizeof(unsigned char));
    cutFrame<<<numBlocksSplit, blockSize>>>(width, height, dataRGB, dataRGBLeft, dataRGBRight);
    hipDeviceSynchronize();
    
    // -- seams definitions
    int* seamsLeft;
    int* seamsRight;
    hipMalloc((void**)&seamsLeft, height*nbAdd*sizeof(int));
    hipMalloc((void**)&seamsRight, height*nbAdd*sizeof(int));
    hipMemcpy(seamsLeft, seamsInLeft, height*nbAdd*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(seamsRight, seamsInRight, height*nbAdd*sizeof(int), hipMemcpyHostToDevice);

    // -- repercute offset of previous seam to all next seams
    int numBlocksReper = (height + blockSize - 1) / blockSize;
    repercuteSeam<<<numBlocksReper, blockSize, 0, streamLeft>>>(nbAdd, seamsLeft, height);
    repercuteSeam<<<numBlocksReper, blockSize, 0, streamRight>>>(nbAdd, seamsRight, height);
    hipDeviceSynchronize();

    // -- add new seams in order
    unsigned char* dataOutDeviceLeft;
    unsigned char* dataOutDevice2Left;
    unsigned char* dataOutDeviceRight;
    unsigned char* dataOutDevice2Right;
    hipMalloc(&dataOutDeviceLeft, widthDst/2*height*3*sizeof(unsigned char));
    hipMalloc(&dataOutDeviceRight, widthDst/2*height*3*sizeof(unsigned char));
    hipMalloc(&dataOutDevice2Left, widthDst/2*height*3*sizeof(unsigned char));
    hipMalloc(&dataOutDevice2Right, widthDst/2*height*3*sizeof(unsigned char));
    hipMemcpy(dataOutDeviceLeft, dataRGBLeft, widthHalf*height*3*sizeof(unsigned char), hipMemcpyDeviceToDevice);
    hipMemcpy(dataOutDeviceRight, dataRGBRight, widthHalf*height*3*sizeof(unsigned char), hipMemcpyDeviceToDevice);

    int widthTemp = widthHalf+1;
    for (int i = 0; i < nbAdd; i++) {
        int numBlocksAdd = ((widthTemp*height*3) + blockSize - 1) / blockSize;
        if (i%2 == 0) {
            addSeamToImage<<<numBlocksAdd, blockSize, 0, streamLeft>>>(widthTemp, height, dataOutDeviceLeft, dataOutDevice2Left, &seamsLeft[i*height]);
            addSeamToImage<<<numBlocksAdd, blockSize, 0, streamRight>>>(widthTemp, height, dataOutDeviceRight, dataOutDevice2Right, &seamsRight[i*height]);
        }
        else {
            addSeamToImage<<<numBlocksAdd, blockSize, 0, streamLeft>>>(widthTemp, height, dataOutDevice2Left, dataOutDeviceLeft, &seamsLeft[i*height]);
            addSeamToImage<<<numBlocksAdd, blockSize, 0, streamRight>>>(widthTemp, height, dataOutDevice2Right, dataOutDeviceRight, &seamsRight[i*height]);
        }
        hipDeviceSynchronize();
        widthTemp++;
    }

    unsigned char* dataOutDevice;
    hipMalloc((void **)&dataOutDevice, widthDst*height*3*sizeof(unsigned char));
    int numBlocksCopy = ((widthDst*height*3) + blockSize - 1) / blockSize;
    if (nbAdd%2 == 0) {copyImage<<<numBlocksCopy,blockSize>>>(widthDst, height, dataOutDeviceLeft, dataOutDeviceRight, dataOutDevice);}
    else {copyImage<<<numBlocksCopy,blockSize>>>(widthDst, height, dataOutDevice2Left, dataOutDevice2Right, dataOutDevice);}
    hipDeviceSynchronize();

    // -- copy final image
    hipMemcpy(dataOut, dataOutDevice, widthDst*height*3*sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipFree(dataRGB);
    hipFree(dataRGBLeft);
    hipFree(dataRGBRight);
    hipFree(dataOutDevice);
    hipFree(dataOutDeviceLeft);
    hipFree(dataOutDeviceRight);
    hipFree(dataOutDevice2Left);
    hipFree(dataOutDevice2Right);
    hipFree(seamsLeft);
    hipFree(seamsRight);
    hipStreamDestroy(streamLeft);
    hipStreamDestroy(streamRight);
}

void addResizeVideoMotionFirst(int width, int height, int widthDst, unsigned char* dataIn, unsigned char* dataOut, int* seamsOutLeft, int* seamsOutRight)
{   
    int blockSize = 1024;
    int widthHalf = width/2;
    int nbAdd = (widthDst-width)/2;
    int searchWindow = 460*widthHalf/1024;
    int leftBoundLeft = 0;
    int rightBoundLeft = searchWindow-1;
    int leftBoundRight = widthHalf - searchWindow;
    int rightBoundRight = widthHalf-1;

    // -- create streams
    hipStream_t streamLeft, streamRight;
    hipStreamCreate(&streamLeft);
    hipStreamCreate(&streamRight);

    // -- copy datas to GPU
    unsigned char* dataRGB;
    hipMalloc(&dataRGB, width*height*3*sizeof(unsigned char));
    hipMemcpy(dataRGB, dataIn, width*height*3*sizeof(unsigned char), hipMemcpyHostToDevice);

    // -- separate left and right
    unsigned char* dataRGBLeft;
    unsigned char* dataRGBRight;
    int numBlocksSplit = ((width*height*3) + blockSize - 1) / blockSize;
    hipMalloc(&dataRGBLeft, widthHalf*height*3*sizeof(unsigned char));
    hipMalloc(&dataRGBRight, widthHalf*height*3*sizeof(unsigned char));
    cutFrame<<<numBlocksSplit, blockSize>>>(width, height, dataRGB, dataRGBLeft, dataRGBRight);
    hipDeviceSynchronize();

    // -- calculate grayscale image
    float* grayImageLeft;
    float* grayImageRight;
    int numBlocksFilter = ((widthHalf*height) + blockSize - 1) / blockSize;
    hipMalloc((void **)&grayImageLeft, widthHalf*height*sizeof(float));
    hipMalloc((void **)&grayImageRight, widthHalf*height*sizeof(float));
    grayscale<<<numBlocksFilter, blockSize, 0, streamLeft>>>(widthHalf, height, dataRGBLeft, grayImageLeft);
    grayscale<<<numBlocksFilter, blockSize, 0, streamRight>>>(widthHalf, height, dataRGBRight, grayImageRight);
    hipDeviceSynchronize();

    // -- calculate sobel
    float* sobelImageLeft;
    float* sobelImageRight;
    hipMalloc((void **)&sobelImageLeft, widthHalf*height*sizeof(float));
    hipMalloc((void **)&sobelImageRight, widthHalf*height*sizeof(float));
    float* sobelImage2Left;
    float* sobelImage2Right;
    hipMalloc((void **)&sobelImage2Left, widthHalf*height*sizeof(float));
    hipMalloc((void **)&sobelImage2Right, widthHalf*height*sizeof(float));
    // -- energy definition
    sobelSimple<<<numBlocksFilter, blockSize, 0, streamLeft>>>(widthHalf, height, grayImageLeft, sobelImageLeft);
    sobelSimple<<<numBlocksFilter, blockSize, 0, streamRight>>>(widthHalf, height, grayImageRight, sobelImageRight);
    hipDeviceSynchronize();

    // -- indexes map definition
    int* indexesLeft;
    int* indexesRight;
    hipMalloc((void**)&indexesLeft, height*widthHalf*sizeof(int));
    hipMalloc((void**)&indexesRight, height*widthHalf*sizeof(int));
    // -- seams definitions
    int* seamsLeft;
    int* seamsRight;
    hipMalloc((void**)&seamsLeft, height*nbAdd*sizeof(int));
    hipMalloc((void**)&seamsRight, height*nbAdd*sizeof(int));

    // -- for other calculation of seams
    int* seamsAllLeft;
    int* seamsAllRight;
    hipMalloc((void**)&seamsAllLeft, height*searchWindow*sizeof(int));
    hipMalloc((void**)&seamsAllRight, height*searchWindow*sizeof(int));
    float* energySeamsLeft;
    float* energySeamsRight;
    hipMalloc((void**)&energySeamsLeft, searchWindow*sizeof(float));
    hipMalloc((void**)&energySeamsRight, searchWindow*sizeof(float));

    // -- start simulation removing
    int widthTemp = widthHalf;
    for (int i = 0; i < nbAdd; i++) {

        float* sobelInLeft;
        float* sobelOutLeft;
        float* sobelInRight;
        float* sobelOutRight;

        // -- alternate between
        if (i%2 == 0) {
            sobelInLeft = sobelImageLeft;
            sobelOutLeft = sobelImage2Left;
            sobelInRight = sobelImageRight;
            sobelOutRight = sobelImage2Right;
        }
        else {
            sobelInLeft = sobelImage2Left;
            sobelOutLeft = sobelImageLeft;
            sobelInRight = sobelImage2Right;
            sobelOutRight = sobelImageRight;
        }

        // -- index map calculation
        hipMemset(indexesLeft, 0, height*widthTemp*sizeof(int));
        hipMemset(indexesRight, 0, height*widthTemp*sizeof(int));
        int numBlocksEnergy = ((widthTemp*(height-1)) + blockSize - 1) / blockSize;
        computeIndexMap<<<numBlocksEnergy, blockSize, 0, streamLeft>>>(widthTemp, height, sobelInLeft, indexesLeft);
        computeIndexMap<<<numBlocksEnergy, blockSize, 0, streamRight>>>(widthTemp, height, sobelInRight, indexesRight);
        hipDeviceSynchronize();

        int numBlocksSeams = ((rightBoundLeft - leftBoundLeft + 1) + blockSize - 1) / blockSize;
        calculateAllSeams<<<numBlocksSeams, blockSize, 0, streamLeft>>>(widthTemp, height, indexesLeft, sobelInLeft, seamsAllLeft, energySeamsLeft, leftBoundLeft, rightBoundLeft);
        calculateAllSeams<<<numBlocksSeams, blockSize, 0, streamRight>>>(widthTemp, height, indexesRight, sobelInRight, seamsAllRight, energySeamsRight, leftBoundRight, rightBoundRight);
        hipDeviceSynchronize();

        keepFirstMotionSeamLeft<<<1,1, 0, streamLeft>>>(height, seamsAllLeft, energySeamsLeft, &seamsLeft[i*height], rightBoundLeft - leftBoundLeft + 1);
        keepFirstMotionSeamRight<<<1,1, 0, streamRight>>>(height, seamsAllRight, energySeamsRight, &seamsRight[i*height], rightBoundRight - leftBoundRight + 1);
        hipDeviceSynchronize();

        // -- compute new sobel
        widthTemp--;
        rightBoundLeft--;
        rightBoundRight--;

        // -- remove seam from sobel
        int numBlocksRemove = ((widthTemp*height) + blockSize - 1) / blockSize;
        removeSeamFromSobel<<<numBlocksRemove, blockSize, 0, streamLeft>>>(widthTemp, height, sobelInLeft, sobelOutLeft, &seamsLeft[i*height]);
        removeSeamFromSobel<<<numBlocksRemove, blockSize, 0, streamRight>>>(widthTemp, height, sobelInRight, sobelOutRight, &seamsRight[i*height]);
        hipDeviceSynchronize();
    }

    hipMemcpy(seamsOutLeft, seamsLeft, height*nbAdd*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(seamsOutRight, seamsRight, height*nbAdd*sizeof(int), hipMemcpyDeviceToHost);

    // -- repercute offset of previous seam to all next seams
    int numBlocksReper = (height + blockSize - 1) / blockSize;
    repercuteSeam<<<numBlocksReper, blockSize, 0, streamLeft>>>(nbAdd, seamsLeft, height);
    repercuteSeam<<<numBlocksReper, blockSize, 0, streamRight>>>(nbAdd, seamsRight, height);
    hipDeviceSynchronize();

    // Free memory
    hipFree(grayImageLeft);
    hipFree(grayImageRight);

    hipFree(sobelImageLeft);
    hipFree(sobelImageRight);
    hipFree(sobelImage2Left);
    hipFree(sobelImage2Right);

    hipFree(indexesLeft);
    hipFree(indexesRight);
    hipFree(seamsAllLeft);
    hipFree(seamsAllRight);
    hipFree(energySeamsLeft);
    hipFree(energySeamsRight);

    // -- add new seams in order
    unsigned char* dataOutDeviceLeft;
    unsigned char* dataOutDevice2Left;
    unsigned char* dataOutDeviceRight;
    unsigned char* dataOutDevice2Right;
    hipMalloc(&dataOutDeviceLeft, widthDst/2*height*3*sizeof(unsigned char));
    hipMalloc(&dataOutDeviceRight, widthDst/2*height*3*sizeof(unsigned char));
    hipMalloc(&dataOutDevice2Left, widthDst/2*height*3*sizeof(unsigned char));
    hipMalloc(&dataOutDevice2Right, widthDst/2*height*3*sizeof(unsigned char));
    hipMemcpy(dataOutDeviceLeft, dataRGBLeft, widthHalf*height*3*sizeof(unsigned char), hipMemcpyDeviceToDevice);
    hipMemcpy(dataOutDeviceRight, dataRGBRight, widthHalf*height*3*sizeof(unsigned char), hipMemcpyDeviceToDevice);

    widthTemp = widthHalf+1;
    for (int i = 0; i < nbAdd; i++) {
        int numBlocksAdd = ((widthTemp*height*3) + blockSize - 1) / blockSize;
        if (i%2 == 0) {
            addSeamToImage<<<numBlocksAdd, blockSize, 0, streamLeft>>>(widthTemp, height, dataOutDeviceLeft, dataOutDevice2Left, &seamsLeft[i*height]);
            addSeamToImage<<<numBlocksAdd, blockSize, 0, streamRight>>>(widthTemp, height, dataOutDeviceRight, dataOutDevice2Right, &seamsRight[i*height]);
        }
        else {
            addSeamToImage<<<numBlocksAdd, blockSize, 0, streamLeft>>>(widthTemp, height, dataOutDevice2Left, dataOutDeviceLeft, &seamsLeft[i*height]);
            addSeamToImage<<<numBlocksAdd, blockSize, 0, streamRight>>>(widthTemp, height, dataOutDevice2Right, dataOutDeviceRight, &seamsRight[i*height]);
        }
        hipDeviceSynchronize();
        widthTemp++;
    }

    unsigned char* dataOutDevice;
    hipMalloc((void **)&dataOutDevice, widthDst*height*3*sizeof(unsigned char));
    int numBlocksCopy = ((widthDst*height*3) + blockSize - 1) / blockSize;
    if (nbAdd%2 == 0) {copyImage<<<numBlocksCopy,blockSize>>>(widthDst, height, dataOutDeviceLeft, dataOutDeviceRight, dataOutDevice);}
    else {copyImage<<<numBlocksCopy,blockSize>>>(widthDst, height, dataOutDevice2Left, dataOutDevice2Right, dataOutDevice);}
    hipDeviceSynchronize();

    // -- copy final image
    hipMemcpy(dataOut, dataOutDevice, widthDst*height*3*sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipFree(dataRGB);
    hipFree(dataRGBLeft);
    hipFree(dataRGBRight);
    hipFree(dataOutDevice);
    hipFree(dataOutDeviceLeft);
    hipFree(dataOutDeviceRight);
    hipFree(dataOutDevice2Left);
    hipFree(dataOutDevice2Right);
    hipFree(seamsLeft);
    hipFree(seamsRight);
    hipStreamDestroy(streamLeft);
    hipStreamDestroy(streamRight);
}
void addResizeVideoMotionOther(int width, int height, int widthDst, unsigned char* dataIn, unsigned char* dataOut, int* seamsInOutLeft, int* seamsInOutRight)
{   
    int blockSize = 1024;
    int widthHalf = width/2;
    int nbAdd = (widthDst-width)/2;
    int searchWindow = 460*widthHalf/1024;
    int leftBoundLeft = 0;
    int rightBoundLeft = searchWindow-1;
    int leftBoundRight = widthHalf - searchWindow;
    int rightBoundRight = widthHalf-1;

    // -- create streams
    hipStream_t streamLeft, streamRight;
    hipStreamCreate(&streamLeft);
    hipStreamCreate(&streamRight);

    // -- copy datas to GPU
    unsigned char* dataRGB;
    hipMalloc(&dataRGB, width*height*3*sizeof(unsigned char));
    hipMemcpy(dataRGB, dataIn, width*height*3*sizeof(unsigned char), hipMemcpyHostToDevice);

    // -- separate left and right
    unsigned char* dataRGBLeft;
    unsigned char* dataRGBRight;
    int numBlocksSplit = ((width*height*3) + blockSize - 1) / blockSize;
    hipMalloc(&dataRGBLeft, widthHalf*height*3*sizeof(unsigned char));
    hipMalloc(&dataRGBRight, widthHalf*height*3*sizeof(unsigned char));
    cutFrame<<<numBlocksSplit, blockSize>>>(width, height, dataRGB, dataRGBLeft, dataRGBRight);
    hipDeviceSynchronize();

    // -- calculate grayscale image
    float* grayImageLeft;
    float* grayImageRight;
    int numBlocksFilter = ((widthHalf*height) + blockSize - 1) / blockSize;
    hipMalloc((void **)&grayImageLeft, widthHalf*height*sizeof(float));
    hipMalloc((void **)&grayImageRight, widthHalf*height*sizeof(float));
    grayscale<<<numBlocksFilter, blockSize, 0, streamLeft>>>(widthHalf, height, dataRGBLeft, grayImageLeft);
    grayscale<<<numBlocksFilter, blockSize, 0, streamRight>>>(widthHalf, height, dataRGBRight, grayImageRight);
    hipDeviceSynchronize();

    // -- calculate sobel
    float* sobelImageLeft;
    float* sobelImageRight;
    hipMalloc((void **)&sobelImageLeft, widthHalf*height*sizeof(float));
    hipMalloc((void **)&sobelImageRight, widthHalf*height*sizeof(float));
    float* sobelImage2Left;
    float* sobelImage2Right;
    hipMalloc((void **)&sobelImage2Left, widthHalf*height*sizeof(float));
    hipMalloc((void **)&sobelImage2Right, widthHalf*height*sizeof(float));
    // -- energy definition
    sobelSimple<<<numBlocksFilter, blockSize, 0, streamLeft>>>(widthHalf, height, grayImageLeft, sobelImageLeft);
    sobelSimple<<<numBlocksFilter, blockSize, 0, streamRight>>>(widthHalf, height, grayImageRight, sobelImageRight);
    hipDeviceSynchronize();

    // -- indexes map definition
    int* indexesLeft;
    int* indexesRight;
    hipMalloc((void**)&indexesLeft, height*widthHalf*sizeof(int));
    hipMalloc((void**)&indexesRight, height*widthHalf*sizeof(int));
    // -- seams definitions
    int* seamsLeft;
    int* seamsRight;
    hipMalloc((void**)&seamsLeft, height*nbAdd*sizeof(int));
    hipMalloc((void**)&seamsRight, height*nbAdd*sizeof(int));

    // -- for other calculation of seams
    int* seamsAllLeft;
    int* seamsAllRight;
    hipMalloc((void**)&seamsAllLeft, height*searchWindow*sizeof(int));
    hipMalloc((void**)&seamsAllRight, height*searchWindow*sizeof(int));
    float* energySeamsLeft;
    float* energySeamsRight;
    hipMalloc((void**)&energySeamsLeft, searchWindow*sizeof(float));
    hipMalloc((void**)&energySeamsRight, searchWindow*sizeof(float));

    // -- seams definitions
    int* seamsLeftPrevious;
    int* seamsRightPrevious;
    hipMalloc((void**)&seamsLeftPrevious, height*nbAdd*sizeof(int));
    hipMalloc((void**)&seamsRightPrevious, height*nbAdd*sizeof(int));
    hipMemcpy(seamsLeftPrevious, seamsInOutLeft, height*nbAdd*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(seamsRightPrevious, seamsInOutRight, height*nbAdd*sizeof(int), hipMemcpyHostToDevice);

    // -- distance to keep better seam
    int* distanceLeft;
    int* distanceRight;
    hipMalloc((void**)&distanceLeft, searchWindow*sizeof(int));
    hipMalloc((void**)&distanceRight, searchWindow*sizeof(int));

    // -- start simulation removing
    int widthTemp = widthHalf;
    for (int i = 0; i < nbAdd; i++) {

        float* sobelInLeft;
        float* sobelOutLeft;
        float* sobelInRight;
        float* sobelOutRight;

        // -- alternate between
        if (i%2 == 0) {
            sobelInLeft = sobelImageLeft;
            sobelOutLeft = sobelImage2Left;
            sobelInRight = sobelImageRight;
            sobelOutRight = sobelImage2Right;
        }
        else {
            sobelInLeft = sobelImage2Left;
            sobelOutLeft = sobelImageLeft;
            sobelInRight = sobelImage2Right;
            sobelOutRight = sobelImageRight;
        }

        // -- index map calculation
        hipMemset(indexesLeft, 0, height*widthTemp*sizeof(int));
        hipMemset(indexesRight, 0, height*widthTemp*sizeof(int));
        int numBlocksEnergy = ((widthTemp*(height-1)) + blockSize - 1) / blockSize;
        computeIndexMap<<<numBlocksEnergy, blockSize, 0, streamLeft>>>(widthTemp, height, sobelInLeft, indexesLeft);
        computeIndexMap<<<numBlocksEnergy, blockSize, 0, streamRight>>>(widthTemp, height, sobelInRight, indexesRight);
        hipDeviceSynchronize();

        int sizeWindow = rightBoundLeft - leftBoundLeft + 1;
        int numBlocksSeams = (sizeWindow + blockSize - 1) / blockSize;
        calculateAllSeams<<<numBlocksSeams, blockSize, 0, streamLeft>>>(widthTemp, height, indexesLeft, sobelInLeft, seamsAllLeft, energySeamsLeft, leftBoundLeft, rightBoundLeft);
        calculateAllSeams<<<numBlocksSeams, blockSize, 0, streamRight>>>(widthTemp, height, indexesRight, sobelInRight, seamsAllRight, energySeamsRight, leftBoundRight, rightBoundRight);
        hipDeviceSynchronize();

        calculateDistanceToOldSeam<<<numBlocksSeams, blockSize, 0, streamLeft>>>(height, &seamsLeftPrevious[i*height], seamsAllLeft, energySeamsLeft, distanceLeft, sizeWindow);
        calculateDistanceToOldSeam<<<numBlocksSeams, blockSize, 0, streamRight>>>(height, &seamsRightPrevious[i*height], seamsAllRight, energySeamsRight, distanceRight, sizeWindow);
        hipDeviceSynchronize();

        keepCloserSeam<<<1,1, 0, streamLeft>>>(height, seamsAllLeft, distanceLeft, &seamsLeft[i*height], sizeWindow);
        keepCloserSeam<<<1,1, 0, streamRight>>>(height, seamsAllRight, distanceRight, &seamsRight[i*height], sizeWindow);
        hipDeviceSynchronize();

        // -- compute new sobel
        widthTemp--;
        rightBoundLeft--;
        rightBoundRight--;

        // -- remove seam from sobel
        int numBlocksRemove = ((widthTemp*height) + blockSize - 1) / blockSize;
        removeSeamFromSobel<<<numBlocksRemove, blockSize, 0, streamLeft>>>(widthTemp, height, sobelInLeft, sobelOutLeft, &seamsLeft[i*height]);
        removeSeamFromSobel<<<numBlocksRemove, blockSize, 0, streamRight>>>(widthTemp, height, sobelInRight, sobelOutRight, &seamsRight[i*height]);
        hipDeviceSynchronize();
    }

    hipMemcpy(seamsInOutLeft, seamsLeft, height*nbAdd*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(seamsInOutRight, seamsRight, height*nbAdd*sizeof(int), hipMemcpyDeviceToHost);

    // -- repercute offset of previous seam to all next seams
    int numBlocksReper = (height + blockSize - 1) / blockSize;
    repercuteSeam<<<numBlocksReper, blockSize, 0, streamLeft>>>(nbAdd, seamsLeft, height);
    repercuteSeam<<<numBlocksReper, blockSize, 0, streamRight>>>(nbAdd, seamsRight, height);
    hipDeviceSynchronize();

    // Free memory
    hipFree(grayImageLeft);
    hipFree(grayImageRight);

    hipFree(sobelImageLeft);
    hipFree(sobelImageRight);
    hipFree(sobelImage2Left);
    hipFree(sobelImage2Right);

    hipFree(indexesLeft);
    hipFree(indexesRight);
    hipFree(seamsAllLeft);
    hipFree(seamsAllRight);
    hipFree(energySeamsLeft);
    hipFree(energySeamsRight);
    hipFree(distanceLeft);
    hipFree(distanceRight);

    // -- add new seams in order
    unsigned char* dataOutDeviceLeft;
    unsigned char* dataOutDevice2Left;
    unsigned char* dataOutDeviceRight;
    unsigned char* dataOutDevice2Right;
    hipMalloc(&dataOutDeviceLeft, widthDst/2*height*3*sizeof(unsigned char));
    hipMalloc(&dataOutDeviceRight, widthDst/2*height*3*sizeof(unsigned char));
    hipMalloc(&dataOutDevice2Left, widthDst/2*height*3*sizeof(unsigned char));
    hipMalloc(&dataOutDevice2Right, widthDst/2*height*3*sizeof(unsigned char));
    hipMemcpy(dataOutDeviceLeft, dataRGBLeft, widthHalf*height*3*sizeof(unsigned char), hipMemcpyDeviceToDevice);
    hipMemcpy(dataOutDeviceRight, dataRGBRight, widthHalf*height*3*sizeof(unsigned char), hipMemcpyDeviceToDevice);

    widthTemp = widthHalf+1;
    for (int i = 0; i < nbAdd; i++) {
        int numBlocksAdd = ((widthTemp*height*3) + blockSize - 1) / blockSize;
        if (i%2 == 0) {
            addSeamToImage<<<numBlocksAdd, blockSize, 0, streamLeft>>>(widthTemp, height, dataOutDeviceLeft, dataOutDevice2Left, &seamsLeft[i*height]);
            addSeamToImage<<<numBlocksAdd, blockSize, 0, streamRight>>>(widthTemp, height, dataOutDeviceRight, dataOutDevice2Right, &seamsRight[i*height]);
        }
        else {
            addSeamToImage<<<numBlocksAdd, blockSize, 0, streamLeft>>>(widthTemp, height, dataOutDevice2Left, dataOutDeviceLeft, &seamsLeft[i*height]);
            addSeamToImage<<<numBlocksAdd, blockSize, 0, streamRight>>>(widthTemp, height, dataOutDevice2Right, dataOutDeviceRight, &seamsRight[i*height]);
        }
        hipDeviceSynchronize();
        widthTemp++;
    }

    unsigned char* dataOutDevice;
    hipMalloc((void **)&dataOutDevice, widthDst*height*3*sizeof(unsigned char));
    int numBlocksCopy = ((widthDst*height*3) + blockSize - 1) / blockSize;
    if (nbAdd%2 == 0) {copyImage<<<numBlocksCopy,blockSize>>>(widthDst, height, dataOutDeviceLeft, dataOutDeviceRight, dataOutDevice);}
    else {copyImage<<<numBlocksCopy,blockSize>>>(widthDst, height, dataOutDevice2Left, dataOutDevice2Right, dataOutDevice);}
    hipDeviceSynchronize();

    // -- copy final image
    hipMemcpy(dataOut, dataOutDevice, widthDst*height*3*sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipFree(dataRGB);
    hipFree(dataRGBLeft);
    hipFree(dataRGBRight);
    hipFree(dataOutDevice);
    hipFree(dataOutDeviceLeft);
    hipFree(dataOutDeviceRight);
    hipFree(dataOutDevice2Left);
    hipFree(dataOutDevice2Right);
    hipFree(seamsLeft);
    hipFree(seamsRight);
    hipStreamDestroy(streamLeft);
    hipStreamDestroy(streamRight);
}

void addResizeVideoDiffFirst(int width, int height, int widthDst, unsigned char* dataIn, unsigned char* dataOut, float* sobelOutLeft, float* sobelOutRight)
{   
    int blockSize = 1024;
    int widthHalf = width/2;
    int nbAdd = (widthDst-width)/2;
    int searchWindow = 460*widthHalf/1024;
    int leftBoundLeft = 0;
    int rightBoundLeft = searchWindow-1;
    int leftBoundRight = widthHalf - searchWindow;
    int rightBoundRight = widthHalf-1;

    // -- create streams
    hipStream_t streamLeft, streamRight;
    hipStreamCreate(&streamLeft);
    hipStreamCreate(&streamRight);

    // -- copy datas to GPU
    unsigned char* dataRGB;
    hipMalloc(&dataRGB, width*height*3*sizeof(unsigned char));
    hipMemcpy(dataRGB, dataIn, width*height*3*sizeof(unsigned char), hipMemcpyHostToDevice);

    // -- separate left and right
    unsigned char* dataRGBLeft;
    unsigned char* dataRGBRight;
    int numBlocksSplit = ((width*height*3) + blockSize - 1) / blockSize;
    hipMalloc(&dataRGBLeft, widthHalf*height*3*sizeof(unsigned char));
    hipMalloc(&dataRGBRight, widthHalf*height*3*sizeof(unsigned char));
    cutFrame<<<numBlocksSplit, blockSize>>>(width, height, dataRGB, dataRGBLeft, dataRGBRight);
    hipDeviceSynchronize();

    // -- calculate grayscale image
    float* grayImageLeft;
    float* grayImageRight;
    int numBlocksFilter = ((widthHalf*height) + blockSize - 1) / blockSize;
    hipMalloc((void **)&grayImageLeft, widthHalf*height*sizeof(float));
    hipMalloc((void **)&grayImageRight, widthHalf*height*sizeof(float));
    grayscale<<<numBlocksFilter, blockSize, 0, streamLeft>>>(widthHalf, height, dataRGBLeft, grayImageLeft);
    grayscale<<<numBlocksFilter, blockSize, 0, streamRight>>>(widthHalf, height, dataRGBRight, grayImageRight);
    hipDeviceSynchronize();

    // -- calculate sobel
    float* sobelImageLeft;
    float* sobelImageRight;
    hipMalloc((void **)&sobelImageLeft, widthHalf*height*sizeof(float));
    hipMalloc((void **)&sobelImageRight, widthHalf*height*sizeof(float));
    float* sobelImage2Left;
    float* sobelImage2Right;
    hipMalloc((void **)&sobelImage2Left, widthHalf*height*sizeof(float));
    hipMalloc((void **)&sobelImage2Right, widthHalf*height*sizeof(float));
    // -- energy definition
    sobelSimple<<<numBlocksFilter, blockSize, 0, streamLeft>>>(widthHalf, height, grayImageLeft, sobelImageLeft);
    sobelSimple<<<numBlocksFilter, blockSize, 0, streamRight>>>(widthHalf, height, grayImageRight, sobelImageRight);
    hipDeviceSynchronize();

    // -- copy out sobel
    hipMemcpy(sobelOutLeft, sobelImageLeft, widthHalf*height*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(sobelOutRight, sobelImageRight, widthHalf*height*sizeof(float), hipMemcpyDeviceToHost);

    // -- indexes map definition
    int* indexesLeft;
    int* indexesRight;
    hipMalloc((void**)&indexesLeft, height*widthHalf*sizeof(int));
    hipMalloc((void**)&indexesRight, height*widthHalf*sizeof(int));
    // -- seams definitions
    int* seamsLeft;
    int* seamsRight;
    hipMalloc((void**)&seamsLeft, height*nbAdd*sizeof(int));
    hipMalloc((void**)&seamsRight, height*nbAdd*sizeof(int));

    // -- for other calculation of seams
    int* seamsAllLeft;
    int* seamsAllRight;
    hipMalloc((void**)&seamsAllLeft, height*searchWindow*sizeof(int));
    hipMalloc((void**)&seamsAllRight, height*searchWindow*sizeof(int));
    float* energySeamsLeft;
    float* energySeamsRight;
    hipMalloc((void**)&energySeamsLeft, searchWindow*sizeof(float));
    hipMalloc((void**)&energySeamsRight, searchWindow*sizeof(float));

    // -- start simulation removing
    int widthTemp = widthHalf;
    for (int i = 0; i < nbAdd; i++) {

        float* sobelInLeft;
        float* sobelOutLeft;
        float* sobelInRight;
        float* sobelOutRight;

        // -- alternate between
        if (i%2 == 0) {
            sobelInLeft = sobelImageLeft;
            sobelOutLeft = sobelImage2Left;
            sobelInRight = sobelImageRight;
            sobelOutRight = sobelImage2Right;
        }
        else {
            sobelInLeft = sobelImage2Left;
            sobelOutLeft = sobelImageLeft;
            sobelInRight = sobelImage2Right;
            sobelOutRight = sobelImageRight;
        }

        // -- index map calculation
        hipMemset(indexesLeft, 0, height*widthTemp*sizeof(int));
        hipMemset(indexesRight, 0, height*widthTemp*sizeof(int));
        int numBlocksEnergy = ((widthTemp*(height-1)) + blockSize - 1) / blockSize;
        computeIndexMap<<<numBlocksEnergy, blockSize, 0, streamLeft>>>(widthTemp, height, sobelInLeft, indexesLeft);
        computeIndexMap<<<numBlocksEnergy, blockSize, 0, streamRight>>>(widthTemp, height, sobelInRight, indexesRight);
        hipDeviceSynchronize();

        int numBlocksSeams = ((rightBoundLeft - leftBoundLeft + 1) + blockSize - 1) / blockSize;
        calculateAllSeams<<<numBlocksSeams, blockSize, 0, streamLeft>>>(widthTemp, height, indexesLeft, sobelInLeft, seamsAllLeft, energySeamsLeft, leftBoundLeft, rightBoundLeft);
        calculateAllSeams<<<numBlocksSeams, blockSize, 0, streamRight>>>(widthTemp, height, indexesRight, sobelInRight, seamsAllRight, energySeamsRight, leftBoundRight, rightBoundRight);
        hipDeviceSynchronize();

        keepFirstMotionSeamLeft<<<1,1, 0, streamLeft>>>(height, seamsAllLeft, energySeamsLeft, &seamsLeft[i*height], rightBoundLeft - leftBoundLeft + 1);
        keepFirstMotionSeamRight<<<1,1, 0, streamRight>>>(height, seamsAllRight, energySeamsRight, &seamsRight[i*height], rightBoundRight - leftBoundRight + 1);
        hipDeviceSynchronize();

        // -- compute new sobel
        widthTemp--;
        rightBoundLeft--;
        rightBoundRight--;

        // -- remove seam from sobel
        int numBlocksRemove = ((widthTemp*height) + blockSize - 1) / blockSize;
        removeSeamFromSobel<<<numBlocksRemove, blockSize, 0, streamLeft>>>(widthTemp, height, sobelInLeft, sobelOutLeft, &seamsLeft[i*height]);
        removeSeamFromSobel<<<numBlocksRemove, blockSize, 0, streamRight>>>(widthTemp, height, sobelInRight, sobelOutRight, &seamsRight[i*height]);
        hipDeviceSynchronize();
    }

    // -- repercute offset of previous seam to all next seams
    int numBlocksReper = (height + blockSize - 1) / blockSize;
    repercuteSeam<<<numBlocksReper, blockSize, 0, streamLeft>>>(nbAdd, seamsLeft, height);
    repercuteSeam<<<numBlocksReper, blockSize, 0, streamRight>>>(nbAdd, seamsRight, height);
    hipDeviceSynchronize();

    // Free memory
    hipFree(grayImageLeft);
    hipFree(grayImageRight);

    hipFree(sobelImageLeft);
    hipFree(sobelImageRight);
    hipFree(sobelImage2Left);
    hipFree(sobelImage2Right);

    hipFree(indexesLeft);
    hipFree(indexesRight);
    hipFree(seamsAllLeft);
    hipFree(seamsAllRight);
    hipFree(energySeamsLeft);
    hipFree(energySeamsRight);

    // -- add new seams in order
    unsigned char* dataOutDeviceLeft;
    unsigned char* dataOutDevice2Left;
    unsigned char* dataOutDeviceRight;
    unsigned char* dataOutDevice2Right;
    hipMalloc(&dataOutDeviceLeft, widthDst/2*height*3*sizeof(unsigned char));
    hipMalloc(&dataOutDeviceRight, widthDst/2*height*3*sizeof(unsigned char));
    hipMalloc(&dataOutDevice2Left, widthDst/2*height*3*sizeof(unsigned char));
    hipMalloc(&dataOutDevice2Right, widthDst/2*height*3*sizeof(unsigned char));
    hipMemcpy(dataOutDeviceLeft, dataRGBLeft, widthHalf*height*3*sizeof(unsigned char), hipMemcpyDeviceToDevice);
    hipMemcpy(dataOutDeviceRight, dataRGBRight, widthHalf*height*3*sizeof(unsigned char), hipMemcpyDeviceToDevice);

    widthTemp = widthHalf+1;
    for (int i = 0; i < nbAdd; i++) {
        int numBlocksAdd = ((widthTemp*height*3) + blockSize - 1) / blockSize;
        if (i%2 == 0) {
            addSeamToImage<<<numBlocksAdd, blockSize, 0, streamLeft>>>(widthTemp, height, dataOutDeviceLeft, dataOutDevice2Left, &seamsLeft[i*height]);
            addSeamToImage<<<numBlocksAdd, blockSize, 0, streamRight>>>(widthTemp, height, dataOutDeviceRight, dataOutDevice2Right, &seamsRight[i*height]);
        }
        else {
            addSeamToImage<<<numBlocksAdd, blockSize, 0, streamLeft>>>(widthTemp, height, dataOutDevice2Left, dataOutDeviceLeft, &seamsLeft[i*height]);
            addSeamToImage<<<numBlocksAdd, blockSize, 0, streamRight>>>(widthTemp, height, dataOutDevice2Right, dataOutDeviceRight, &seamsRight[i*height]);
        }
        hipDeviceSynchronize();
        widthTemp++;
    }

    unsigned char* dataOutDevice;
    hipMalloc((void **)&dataOutDevice, widthDst*height*3*sizeof(unsigned char));
    int numBlocksCopy = ((widthDst*height*3) + blockSize - 1) / blockSize;
    if (nbAdd%2 == 0) {copyImage<<<numBlocksCopy,blockSize>>>(widthDst, height, dataOutDeviceLeft, dataOutDeviceRight, dataOutDevice);}
    else {copyImage<<<numBlocksCopy,blockSize>>>(widthDst, height, dataOutDevice2Left, dataOutDevice2Right, dataOutDevice);}
    hipDeviceSynchronize();

    // -- copy final image
    hipMemcpy(dataOut, dataOutDevice, widthDst*height*3*sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipFree(dataRGB);
    hipFree(dataRGBLeft);
    hipFree(dataRGBRight);
    hipFree(dataOutDevice);
    hipFree(dataOutDeviceLeft);
    hipFree(dataOutDeviceRight);
    hipFree(dataOutDevice2Left);
    hipFree(dataOutDevice2Right);
    hipFree(seamsLeft);
    hipFree(seamsRight);
    hipStreamDestroy(streamLeft);
    hipStreamDestroy(streamRight);
}
void addResizeVideoDiffOther(int width, int height, int widthDst, unsigned char* dataIn, unsigned char* dataOut, float* sobelInOutLeft, float* sobelInOutRight)
{   
    int blockSize = 1024;
    int widthHalf = width/2;
    int nbAdd = (widthDst-width)/2;
    int searchWindow = 460*widthHalf/1024;
    int leftBoundLeft = 0;
    int rightBoundLeft = searchWindow-1;
    int leftBoundRight = widthHalf - searchWindow;
    int rightBoundRight = widthHalf-1;

    // -- create streams
    hipStream_t streamLeft, streamRight;
    hipStreamCreate(&streamLeft);
    hipStreamCreate(&streamRight);

    // -- copy datas to GPU
    unsigned char* dataRGB;
    hipMalloc(&dataRGB, width*height*3*sizeof(unsigned char));
    hipMemcpy(dataRGB, dataIn, width*height*3*sizeof(unsigned char), hipMemcpyHostToDevice);

    // -- separate left and right
    unsigned char* dataRGBLeft;
    unsigned char* dataRGBRight;
    int numBlocksSplit = ((width*height*3) + blockSize - 1) / blockSize;
    hipMalloc(&dataRGBLeft, widthHalf*height*3*sizeof(unsigned char));
    hipMalloc(&dataRGBRight, widthHalf*height*3*sizeof(unsigned char));
    cutFrame<<<numBlocksSplit, blockSize>>>(width, height, dataRGB, dataRGBLeft, dataRGBRight);
    hipDeviceSynchronize();

    // -- calculate grayscale image
    float* grayImageLeft;
    float* grayImageRight;
    int numBlocksFilter = ((widthHalf*height) + blockSize - 1) / blockSize;
    hipMalloc((void **)&grayImageLeft, widthHalf*height*sizeof(float));
    hipMalloc((void **)&grayImageRight, widthHalf*height*sizeof(float));
    grayscale<<<numBlocksFilter, blockSize, 0, streamLeft>>>(widthHalf, height, dataRGBLeft, grayImageLeft);
    grayscale<<<numBlocksFilter, blockSize, 0, streamRight>>>(widthHalf, height, dataRGBRight, grayImageRight);
    hipDeviceSynchronize();

    // -- calculate sobel
    float* sobelImageLeft;
    float* sobelImageRight;
    hipMalloc((void **)&sobelImageLeft, widthHalf*height*sizeof(float));
    hipMalloc((void **)&sobelImageRight, widthHalf*height*sizeof(float));
    float* sobelImage2Left;
    float* sobelImage2Right;
    hipMalloc((void **)&sobelImage2Left, widthHalf*height*sizeof(float));
    hipMalloc((void **)&sobelImage2Right, widthHalf*height*sizeof(float));
    hipMemcpy(sobelImage2Left, sobelInOutLeft, widthHalf*height*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(sobelImage2Right, sobelInOutRight, widthHalf*height*sizeof(float), hipMemcpyHostToDevice);

    // -- energy definition
    sobelComplex<<<numBlocksFilter, blockSize, 0, streamLeft>>>(widthHalf, height, grayImageLeft, sobelImage2Left, sobelImageLeft);
    sobelComplex<<<numBlocksFilter, blockSize, 0, streamRight>>>(widthHalf, height, grayImageRight, sobelImage2Right, sobelImageRight);
    hipDeviceSynchronize();

    // -- copy out sobel
    hipMemcpy(sobelInOutLeft, sobelImageLeft, widthHalf*height*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(sobelInOutRight, sobelImageRight, widthHalf*height*sizeof(float), hipMemcpyDeviceToHost);
    //hipMemcpy(sobelInOutLeft, sobelImage2Left, widthHalf*height*sizeof(float), hipMemcpyDeviceToHost);
    //hipMemcpy(sobelInOutRight, sobelImage2Left, widthHalf*height*sizeof(float), hipMemcpyDeviceToHost);

    // -- indexes map definition
    int* indexesLeft;
    int* indexesRight;
    hipMalloc((void**)&indexesLeft, height*widthHalf*sizeof(int));
    hipMalloc((void**)&indexesRight, height*widthHalf*sizeof(int));
    // -- seams definitions
    int* seamsLeft;
    int* seamsRight;
    hipMalloc((void**)&seamsLeft, height*nbAdd*sizeof(int));
    hipMalloc((void**)&seamsRight, height*nbAdd*sizeof(int));

    // -- for other calculation of seams
    int* seamsAllLeft;
    int* seamsAllRight;
    hipMalloc((void**)&seamsAllLeft, height*searchWindow*sizeof(int));
    hipMalloc((void**)&seamsAllRight, height*searchWindow*sizeof(int));
    float* energySeamsLeft;
    float* energySeamsRight;
    hipMalloc((void**)&energySeamsLeft, searchWindow*sizeof(float));
    hipMalloc((void**)&energySeamsRight, searchWindow*sizeof(float));

    // -- start simulation removing
    int widthTemp = widthHalf;
    for (int i = 0; i < nbAdd; i++) {

        float* sobelInLeft;
        float* sobelOutLeft;
        float* sobelInRight;
        float* sobelOutRight;

        // -- alternate between
        if (i%2 == 0) {
            sobelInLeft = sobelImageLeft;
            sobelOutLeft = sobelImage2Left;
            sobelInRight = sobelImageRight;
            sobelOutRight = sobelImage2Right;
        }
        else {
            sobelInLeft = sobelImage2Left;
            sobelOutLeft = sobelImageLeft;
            sobelInRight = sobelImage2Right;
            sobelOutRight = sobelImageRight;
        }

        // -- index map calculation
        hipMemset(indexesLeft, 0, height*widthTemp*sizeof(int));
        hipMemset(indexesRight, 0, height*widthTemp*sizeof(int));
        int numBlocksEnergy = ((widthTemp*(height-1)) + blockSize - 1) / blockSize;
        computeIndexMap<<<numBlocksEnergy, blockSize, 0, streamLeft>>>(widthTemp, height, sobelInLeft, indexesLeft);
        computeIndexMap<<<numBlocksEnergy, blockSize, 0, streamRight>>>(widthTemp, height, sobelInRight, indexesRight);
        hipDeviceSynchronize();

        int numBlocksSeams = ((rightBoundLeft - leftBoundLeft + 1) + blockSize - 1) / blockSize;
        calculateAllSeams<<<numBlocksSeams, blockSize, 0, streamLeft>>>(widthTemp, height, indexesLeft, sobelInLeft, seamsAllLeft, energySeamsLeft, leftBoundLeft, rightBoundLeft);
        calculateAllSeams<<<numBlocksSeams, blockSize, 0, streamRight>>>(widthTemp, height, indexesRight, sobelInRight, seamsAllRight, energySeamsRight, leftBoundRight, rightBoundRight);
        hipDeviceSynchronize();

        keepFirstMotionSeamLeft<<<1,1, 0, streamLeft>>>(height, seamsAllLeft, energySeamsLeft, &seamsLeft[i*height], rightBoundLeft - leftBoundLeft + 1);
        keepFirstMotionSeamRight<<<1,1, 0, streamRight>>>(height, seamsAllRight, energySeamsRight, &seamsRight[i*height], rightBoundRight - leftBoundRight + 1);
        hipDeviceSynchronize();

        // -- compute new sobel
        widthTemp--;
        rightBoundLeft--;
        rightBoundRight--;

        // -- remove seam from sobel
        int numBlocksRemove = ((widthTemp*height) + blockSize - 1) / blockSize;
        removeSeamFromSobel<<<numBlocksRemove, blockSize, 0, streamLeft>>>(widthTemp, height, sobelInLeft, sobelOutLeft, &seamsLeft[i*height]);
        removeSeamFromSobel<<<numBlocksRemove, blockSize, 0, streamRight>>>(widthTemp, height, sobelInRight, sobelOutRight, &seamsRight[i*height]);
        hipDeviceSynchronize();
    }

    // -- repercute offset of previous seam to all next seams
    int numBlocksReper = (height + blockSize - 1) / blockSize;
    repercuteSeam<<<numBlocksReper, blockSize, 0, streamLeft>>>(nbAdd, seamsLeft, height);
    repercuteSeam<<<numBlocksReper, blockSize, 0, streamRight>>>(nbAdd, seamsRight, height);
    hipDeviceSynchronize();

    // Free memory
    hipFree(grayImageLeft);
    hipFree(grayImageRight);

    hipFree(sobelImageLeft);
    hipFree(sobelImageRight);
    hipFree(sobelImage2Left);
    hipFree(sobelImage2Right);

    hipFree(indexesLeft);
    hipFree(indexesRight);
    hipFree(seamsAllLeft);
    hipFree(seamsAllRight);
    hipFree(energySeamsLeft);
    hipFree(energySeamsRight);

    // -- add new seams in order
    unsigned char* dataOutDeviceLeft;
    unsigned char* dataOutDevice2Left;
    unsigned char* dataOutDeviceRight;
    unsigned char* dataOutDevice2Right;
    hipMalloc(&dataOutDeviceLeft, widthDst/2*height*3*sizeof(unsigned char));
    hipMalloc(&dataOutDeviceRight, widthDst/2*height*3*sizeof(unsigned char));
    hipMalloc(&dataOutDevice2Left, widthDst/2*height*3*sizeof(unsigned char));
    hipMalloc(&dataOutDevice2Right, widthDst/2*height*3*sizeof(unsigned char));
    hipMemcpy(dataOutDeviceLeft, dataRGBLeft, widthHalf*height*3*sizeof(unsigned char), hipMemcpyDeviceToDevice);
    hipMemcpy(dataOutDeviceRight, dataRGBRight, widthHalf*height*3*sizeof(unsigned char), hipMemcpyDeviceToDevice);

    widthTemp = widthHalf+1;
    for (int i = 0; i < nbAdd; i++) {
        int numBlocksAdd = ((widthTemp*height*3) + blockSize - 1) / blockSize;
        if (i%2 == 0) {
            addSeamToImage<<<numBlocksAdd, blockSize, 0, streamLeft>>>(widthTemp, height, dataOutDeviceLeft, dataOutDevice2Left, &seamsLeft[i*height]);
            addSeamToImage<<<numBlocksAdd, blockSize, 0, streamRight>>>(widthTemp, height, dataOutDeviceRight, dataOutDevice2Right, &seamsRight[i*height]);
        }
        else {
            addSeamToImage<<<numBlocksAdd, blockSize, 0, streamLeft>>>(widthTemp, height, dataOutDevice2Left, dataOutDeviceLeft, &seamsLeft[i*height]);
            addSeamToImage<<<numBlocksAdd, blockSize, 0, streamRight>>>(widthTemp, height, dataOutDevice2Right, dataOutDeviceRight, &seamsRight[i*height]);
        }
        hipDeviceSynchronize();
        widthTemp++;
    }

    unsigned char* dataOutDevice;
    hipMalloc((void **)&dataOutDevice, widthDst*height*3*sizeof(unsigned char));
    int numBlocksCopy = ((widthDst*height*3) + blockSize - 1) / blockSize;
    if (nbAdd%2 == 0) {copyImage<<<numBlocksCopy,blockSize>>>(widthDst, height, dataOutDeviceLeft, dataOutDeviceRight, dataOutDevice);}
    else {copyImage<<<numBlocksCopy,blockSize>>>(widthDst, height, dataOutDevice2Left, dataOutDevice2Right, dataOutDevice);}
    hipDeviceSynchronize();

    // -- copy final image
    hipMemcpy(dataOut, dataOutDevice, widthDst*height*3*sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipFree(dataRGB);
    hipFree(dataRGBLeft);
    hipFree(dataRGBRight);
    hipFree(dataOutDevice);
    hipFree(dataOutDeviceLeft);
    hipFree(dataOutDeviceRight);
    hipFree(dataOutDevice2Left);
    hipFree(dataOutDevice2Right);
    hipFree(seamsLeft);
    hipFree(seamsRight);
    hipStreamDestroy(streamLeft);
    hipStreamDestroy(streamRight);
}